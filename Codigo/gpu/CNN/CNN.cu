#include "hip/hip_runtime.h"
#include "CNN.h"

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }else
    {
        cout << "Todo correcto!" << endl;
    }
}


/*
    CONSTRUCTOR de la clase CNN
    --------------------------------------
  
    @capas_conv     Indica el número de capas convolucionales, así como la estructura de cada una. Habrá "capas_conv.size()" capas convolucionales, y la estructura de la capa i 
                    vendrá dada por capas_conv[i]. De esta forma. capas_conv[i] = {3, 2, 2} corresponde a un kernel 3x2x2, por ejemplo.
    @tams_pool      Indica el número de capas de agrupación, así como la estructura de cada una. tams_pool[i] = {2,2} indica un tamaño de ventana de agrupamiento  de 2x2.
    @padding        Indica el nivel de padding de cada capa convolucional. padding[i] corresponde con el nivel de padding a aplicar en la capa capas_conv[i].
    @capas_fully    Vector de enteros que indica el número de neuronas por capa dentro de la capa totalmente conectada. Habrá capas.size() capas y cada una contendrá capas[i] neuronas.
    @input          Volumen 3D de entrada. Se tendrán en cuenta sus dimensiones para crear las estructuras necesarias y permitir un posterior entrenamiento con volúmenes de iguales dimensiones.
    @lr             Learning Rate o Tasa de Aprendizaje
*/
CNN::CNN(int *capas_conv, int n_capas_conv, int *tams_pool, int *padding, int *capas_fully, int n_capas_fully, int C, int H, int W, const float &lr)
{
    int * i_capas_conv = nullptr;
    int * i_capas_pool = nullptr;

    // Ejemplo de uso, capas_conv[0] = {16, 3, 3}

    if(C <= 0)
    {
        cout << "ERROR. Hay que proporcionar un input a la red. \n";
        exit(-1);
    }

    this->n_capas_conv = n_capas_conv;
    this->lr = lr;
    this->convs = new Convolutional[this->n_capas_conv];
    this->plms = new PoolingMax[this->n_capas_conv];
    this->padding = (int *)malloc(n_capas_conv * sizeof(int));
    this->n_clases = capas_fully[n_capas_fully-1];
    this->max_H = 0;
    this->max_W = 0;
    this->max_C = 0;
    this->i_conv_out = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_conv_in = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_plm_out = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_plm_in = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_w = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_b = (int *)malloc(n_capas_conv * sizeof(int));

    for(int i=0; i<n_capas_conv; i++)
        this->padding[i] = padding[i];

    // Padding de la primera capa
    H += 2*padding[0];
    W += 2*padding[0];
    
    if(max_C < C)
        max_C = C;

    if(max_H < H)
        max_H = H;
    
    if(max_W < W)
        max_W = W;

    // Inicializar capas convolucionales y maxpool --------------------------------------------
    for(int i=0; i<n_capas_conv; ++i)
    {   
        i_capas_conv = capas_conv + 3*i;
        i_capas_pool = tams_pool + 2*i;

        // Capas convolucionales ------------------------------------------------
        //                  nºkernels          filas_kernel      cols_kernel
        Convolutional conv(i_capas_conv[0], i_capas_conv[1], i_capas_conv[2], C, H, W, lr);
        this->convs[i].copiar(conv);

        // H_out = H - K + 1
        C = i_capas_conv[0];
        H = H - i_capas_conv[1] + 1;
        W = W - i_capas_conv[2] + 1;

        if(max_C < C)
            max_C = C;

        if(max_H < H)
            max_H = H;
        
        if(max_W < W)
            max_W = W;

        // Capas MaxPool -----------------------------------------------------------
        int pad_sig = 0;    // Padding de la siguiente capa convolucional
        if(this->n_capas_conv > i+1)
            pad_sig = this->padding[i+1];
        //           filas_kernel_pool  cols_kernel_pool
        PoolingMax plm(i_capas_pool[0], i_capas_pool[1], C, H, W, pad_sig);
        this->plms[i].copiar(plm);

        // H_out = H / K + 2*pad
        H = H / i_capas_pool[0] + 2*pad_sig;
        W = W / i_capas_pool[0] + 2*pad_sig;

        if(max_C < C)
            max_C = C;

        if(max_H < H)
            max_H = H;
        
        if(max_W < W)
            max_W = W;
    }

    
    // Inicializar capa fullyconnected -----------------------------------------
    int *capas_fully_ptr = (int *)malloc((n_capas_fully+1) * sizeof(int));

    capas_fully_ptr[0] = C*H*W;

    for(int i=1; i<n_capas_fully+1; i++)
        capas_fully_ptr[i] = capas_fully[i-1];

    this->fully = new FullyConnected(capas_fully_ptr, n_capas_fully+1, lr);

    // Reserva de espacio para posteriores operaciones
    int tam_img_max = max_C*max_H*max_W;
    //int tam_img_max = max_C*max_H*max_W;

    this->img_in = (float *)malloc(tam_img_max * sizeof(float));
    this->img_out = (float *)malloc(tam_img_max * sizeof(float)); 
    this->img_in_copy = (float *)malloc(tam_img_max * sizeof(float)); 
    this->conv_a = (float *)malloc(tam_img_max * sizeof(float));
    this->a_ptr = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float));
    this->z_ptr = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float)); 

    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------
    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------
    for(int i=0; i<tam_img_max; i++)
    {
        this->img_in[i] = 1.0;
        this->img_out[i] = 2.0;
        this->conv_a[i] = 3.0;
    }
    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------
    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------

    int i_out_c = 0, i_in_c = 0, i_out_p = 0, i_in_p = 0, i_w_ = 0, i_b_ = 0;
    for(int i=0; i<n_capas_conv-1; i++)
    {
        // Convolucional
        this->i_conv_in[i] = i_in_c;
        i_in_c += this->convs[i].get_C() * this->convs[i].get_H() * this->convs[i].get_W(); 

        this->i_conv_out[i] = i_out_c;
        i_out_c += this->convs[i].get_n_kernels() * this->convs[i].get_H_out() * this->convs[i].get_W_out();   

        // Agrupación máxima
        this->i_plm_in[i] = i_in_p;
        i_in_p += this->plms[i].get_C() * this->plms[i].get_H() * this->plms[i].get_W(); 

        this->i_plm_out[i] = i_out_p;
        i_out_p += this->plms[i].get_C() * this->plms[i].get_H_out() * this->plms[i].get_W_out();  

        i_w[i] = i_w_;
        i_w_ += this->convs[i].get_n_kernels() * this->convs[i].get_C() * this->convs[i].get_kernel_fils() * this->convs[i].get_kernel_cols();

        i_b[i] = i_b_;
        i_b_ += this->convs[i].get_n_kernels();
    }

    // Liberar memoria
    free(capas_fully_ptr);
}

/*
    @brief  Muestra la arquitectura de la red
*/
void CNN::mostrar_arquitectura()
{
    cout << "\n-----------Arquitectura de la red-----------\n";
    cout << "Padding por capa: ";
    for(int i=0; i<this->n_capas_conv-1; i++)
        cout << this->padding[i] << ", ";
    cout << this->padding[this->n_capas_conv-1];
    cout << endl;
    
    for(int i=0; i<this->n_capas_conv; i++)
    {
        cout << "Dimensiones de entrada a " << this->convs[i].get_n_kernels() << " kernels " << this->convs[i].get_kernel_fils() << "x" << this->convs[i].get_kernel_cols() << " convolucionales: " << this->convs[i].get_C() << "x" << this->convs[i].get_H() << "x" << this->convs[i].get_W() << endl;
        cout << "Dimensiones de entrada a un kernel " << this->plms[i].get_kernel_fils() << "x" << this->plms[i].get_kernel_cols() << " MaxPool: " << this->plms[i].get_C() << "x" << this->plms[i].get_H() << "x" << this->plms[i].get_W() << endl;
    }

    // Volúmen de salida de la última capa MaxPool
    cout << "Dimensiones de salida de un kernel " << this->plms[this->n_capas_conv-1].get_kernel_fils() << "x" << this->plms[this->n_capas_conv-1].get_kernel_cols() << " MaxPool: " << this->plms[this->n_capas_conv-1].get_C() << "x" << this->plms[this->n_capas_conv-1].get_H_out() << "x" << this->plms[this->n_capas_conv-1].get_W_out() << endl;

    // Capas totalmente conectadas
    int * capas = this->fully->get_capas();

    cout << "Capas totalmente concetadas: ";
    for(int i=0; i<this->fully->get_n_capas()-1; i++)
        cout << capas[i] << ", ";
    cout << capas[this->fully->get_n_capas()-1];
    
    cout << endl;
}

void CNN::set_train(float *x, float *y, int n_imgs, int n_clases, int C, int H, int W)
{
    n_imgs -= 1;
    H += 2*this->padding[0];
    W += 2*this->padding[0];
    this->n_imagenes = n_imgs * n_clases;
    this->train_imgs = (float *)malloc(n_imagenes*C*H*W * sizeof(float));
    this->train_labels = (float *)malloc(n_imagenes*n_clases * sizeof(float));

    if(this->n_clases != n_clases)
        cout << "\n\nError. Número de clases distinto al establecido previamente en la arquitectura de la red. " << this->n_clases << " != " << n_clases << endl << endl;

    for(int i=0; i<n_imagenes*C*H*W; i++)
        train_imgs[i] = x[i];
    
    for(int i=0; i<n_imagenes*n_clases; i++)
        train_labels[i] = y[i];

    /*
    // Mostrar imágenes
    cout << "\nX\n";
    for(int i=0; i<this->n_imagenes; i++)
    {
        for(int j=0; j<C; j++)
        {
            for(int k=0; k<H; k++)
            {
                for(int p=0; p<W; p++)
                    cout << x[i*C*H*W + j*H*W + k*W + p] << " ";
                cout << endl;
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << endl;

    cout << "\nY\n";
    for(int i=0; i<this->n_imagenes; i++)
    {
        for(int j=0; j<n_clases; j++)
            cout << y[i*n_clases + j] << " ";
        cout << endl;
    }
    cout << endl;
    */
}

/*
    @brief  Aplica padding sobre una imagen sin aumentar su tamaño
    @input  Imagen sobre la cual aplicar padding
    @pad    Nivel de padding a aplicar
    @return Imagen @input con padding interno aplicado
*/
void CNN::padding_interno_ptr(float *input, int C, int H, int W, const int &pad)
{
    for(int i=0; i<C; ++i)
    {
        // Primeras "pad" filas se igualan a 0.0
        for(int j=0; j<pad; ++j)
            for(int k=0; k<H; ++k)
            input[i*H*W + j*W + k] = 0.0; 

        // Últimas "pad" filas se igualan a 0.0
        for(int j=H-1; j>=H - pad; j--)
            for(int k=0; k<H; ++k)
            input[i*H*W + j*W + k] = 0.0; 
        
        // Por cada fila
        for(int k=0; k<H; ++k)
        {
            // Primeras "pad" casillas se igualan a 0.0
            for(int j=0; j<pad; ++j)
                input[i*H*W + j*W + k] = 0.0;

            // Últimas "pad" casillas se igualan a 0.0
            for(int j=W-1; j>=W - pad; j--)
                input[i*H*W + j*W + k] = 0.0;
        }
    }    
}

/*
    @brief  Aplica padding sobre una imagen sin aumentar su tamaño
    @input  Imagen sobre la cual aplicar padding
    @pad    Nivel de padding a aplicar
    @return Imagen @input con padding interno aplicado
*/
void CNN::padding_interno(vector<vector<vector<float>>> &input, const int &pad)
{
    for(int i=0; i<input.size(); ++i)
    {
        // Primeras "pad" filas se igualan a 0.0
        for(int j=0; j<pad; ++j)
            for(int k=0; k<input[i].size(); ++k)
            input[i][j][k] = 0.0; 

        // Últimas "pad" filas se igualan a 0.0
        for(int j=input[i].size()-1; j>=input[i].size() - pad; j--)
            for(int k=0; k<input[i].size(); ++k)
            input[i][j][k] = 0.0; 
        
        // Por cada fila
        for(int k=0; k<input[i].size(); ++k)
        {
            // Primeras "pad" casillas se igualan a 0.0
            for(int j=0; j<pad; ++j)
                input[i][k][j] = 0.0;

            // Últimas "pad" casillas se igualan a 0.0
            for(int j=input[i][k].size()-1; j>=input[i][k].size() - pad; j--)
                input[i][k][j] = 0.0;
        }
    }    
}


void shuffle(int *vec, int tam_vec, mt19937& rng) {
    for (int i = tam_vec - 1; i > 0; --i) {
        std::uniform_int_distribution<int> dist(0, i);
        int j = dist(rng);
        std::swap(vec[i], vec[j]);
    }
}


/*
                cout << "Input" << endl;
                for(int i=0; i<this->convs[0].get_C(); i++)
                {
                    for(int j=0; j<this->convs[0].get_H(); j++)
                    {
                        for(int k=0; k<this->convs[0].get_W(); k++)
                            cout << img_train[i*this->convs[0].get_H()*this->convs[0].get_W() + j*this->convs[0].get_W() + k] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                int pepe;
                cin >> pepe;

                cout << "Output" << endl;
                for(int i=0; i<this->convs[0].get_n_kernels(); i++)
                {
                    for(int j=0; j<this->convs[0].get_H_out(); j++)
                    {
                        for(int k=0; k<this->convs[0].get_W_out(); k++)
                            cout << img_conv_out[i*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j*this->convs[0].get_W_out() + k] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                cin >> pepe;
*/


void CNN::train(int epocas, int mini_batch)
{
    
    auto ini = high_resolution_clock::now();
    auto fin = high_resolution_clock::now();
    auto duration = duration_cast<seconds>(fin - ini);

    int n=this->n_imagenes;
    int C, H_out, W_out;
    /*
    vector<vector<vector<vector<vector<float>>>>> convs_outs(mini_batch), plms_outs(mini_batch), conv_grads_w(this->n_capas_conv), plms_in_copys(mini_batch), conv_a(mini_batch);       // Input y output de cada capa (por cada imagen de training)
    vector<vector<vector<vector<float>>>> convs_out(this->n_capas_conv), pools_out(this->n_capas_conv);
    vector<vector<vector<float>>> grads_pesos_fully = (*this->fully).get_pesos(), img_aux;
    vector<vector<float>> grad_x_fully, flat_outs(mini_batch), grads_bias_fully = (*this->fully).get_bias(), fully_a = (*this->fully).get_a(), fully_z = fully_a, fully_grad_a = fully_a, conv_grads_bias(this->n_capas_conv), prueba(this->n_capas_conv), max_conv(this->n_capas_conv), min_conv(this->n_capas_conv); 
    */
   
    int tam_in_convs = 0, tam_out_convs = 0, tam_in_pools = 0, tam_out_pools = 0, tam_kernels_conv = 0, 
        tam_flat_out = this->plms[this->n_capas_conv-1].get_C() * this->plms[this->n_capas_conv-1].get_H_out() * this->plms[this->n_capas_conv-1].get_W_out(),
        n_bias_conv = 0;

    for(int i=0; i<this->n_capas_conv; i++)
    {
        tam_kernels_conv += this->convs[i].get_n_kernels() * this->convs[i].get_C() * this->convs[i].get_kernel_fils() * this->convs[i].get_kernel_cols(); 
        tam_in_convs += this->convs[i].get_C() * this->convs[i].get_H() * this->convs[i].get_W(); 
        tam_out_convs += this->convs[i].get_n_kernels() * this->convs[i].get_H_out() * this->convs[i].get_W_out(); 
        tam_out_pools += this->plms[i].get_C() * this->plms[i].get_H_out() * this->plms[i].get_W_out(); 
        tam_in_pools += this->plms[i].get_C() * this->plms[i].get_H() * this->plms[i].get_W(); 
        n_bias_conv += this->convs[i].get_n_kernels();
    }


    float *grad_x_fully = (float *)malloc(mini_batch* this->fully->get_capas()[0] * sizeof(float)),
          *flat_outs = (float *)malloc(mini_batch* tam_flat_out * sizeof(float)),
          *fully_grad_a = (float *)malloc(mini_batch* this->fully->get_n_neuronas() * sizeof(float)),
          *conv_grads_bias = (float *)malloc(n_bias_conv * sizeof(float)),
          *grads_bias_fully = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float)),
          *grads_pesos_fully = (float *)malloc(this->fully->get_n_pesos() * sizeof(float)),
          *convs_outs = (float *)malloc(mini_batch * tam_out_convs * sizeof(float)), 
          *plms_outs = (float *)malloc(mini_batch * tam_out_pools * sizeof(float)), 
          *conv_grads_w = (float *)malloc(tam_kernels_conv * sizeof(float)), 
          *plms_in_copys = (float *)malloc(mini_batch * tam_in_pools* sizeof(float)), 
          *conv_a = (float *)malloc(mini_batch * tam_in_convs * sizeof(float)); 

    float *img_train = nullptr;
    float *img_conv_out = nullptr;
    float *img_conv_a = nullptr;
    float *img_plms_out = nullptr;
    float *img_plms_in_copy = nullptr;
    float *img_flat_out = nullptr;
    float *img_grad_x_fully = nullptr;
    float *img_grad_w_conv = nullptr;
    float *img_grad_b_conv = nullptr;

    const int M = n / mini_batch;
    int pad_sig, C_ini = this->convs[0].get_C(), H_ini = this->convs[0].get_H(), W_ini = this->convs[0].get_W(), tam_ini = C_ini*H_ini*W_ini;

    std::random_device rd;
    std::mt19937 g(rd());

    int n_batches = M;
    if(n % mini_batch != 0)
        n_batches++;
    int *indices = (int *)malloc(n * sizeof(int)),
        *batch = (int *)malloc(mini_batch * sizeof(int)),
        *tam_batches = (int *)malloc(n_batches * sizeof(int));
    
    //-------------------------------------------------
    // Inicializar índices
    //-------------------------------------------------
    // Inicializar vector de índices
    for(int i=0; i<n; ++i)
        indices[i] = i;

    // Inicializar tamaño de mini-batches
    for(int i=0; i<M; ++i)
        tam_batches[i] = mini_batch;
    
    // Último batch puede tener distinto tamaño al resto
    if(n % mini_batch != 0)
        tam_batches[n_batches-1] = n % mini_batch;   


    for(int ep=0; ep<epocas; ++ep)
    {
        ini = high_resolution_clock::now();

        // Desordenar vector de índices
        shuffle(indices, n, g);

        
        // ForwardPropagation de cada batch -----------------------------------------------------------------------
        for(int i=0; i<n_batches; ++i)
        {
            // Crear el batch para cada hebra ----------------------
            for(int j=0; j<tam_batches[i]; j++)
                batch[j] = indices[mini_batch*i + j];   

            
            for(int img=0; img<tam_batches[i]; ++img)
                for(int j=0; j<this->n_capas_conv; ++j)
                {
                    pad_sig = 0;    // Padding de la siguiente capa convolucional
                    if(this->n_capas_conv > j+1)
                        pad_sig = this->padding[j+1];

                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j];
                    padding_interno_ptr(img_plms_out, this->plms[j].get_C(), this->plms[j].get_H_out(), this->plms[j].get_W_out(), pad_sig);
                }
            

            // ---------------------------------------------------------------------------------------
            for(int img=0; img<tam_batches[i]; ++img)
            {
                // Primera capa convolucional y maxpool -----------------------
                img_train = this->train_imgs + tam_ini*batch[img];
                img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[0];
                img_conv_a = conv_a + img*tam_in_convs + i_conv_in[0];
                this->convs[0].forwardPropagationGEMM(img_train, img_conv_out, img_conv_a);

                /*
                cout << "Input" << endl;
                for(int i_=0; i_<this->convs[0].get_C(); i_++)
                {
                    for(int j_=0; j_<this->convs[0].get_H(); j_++)
                    {
                        for(int k_=0; k_<this->convs[0].get_W(); k_++)
                            cout << img_train[i_*this->convs[0].get_H()*this->convs[0].get_W() + j_*this->convs[0].get_W() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                int k1;
                //cin >> k1;

                cout << "Output CV" << endl;
                for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                            cout << img_conv_out[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */

                img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[0];
                img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[0];
                this->plms[0].forwardPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);


                /*
                cout << "Output PM" << endl;
                for(int i_=0; i_<this->plms[0].get_C(); i_++)
                {
                    for(int j_=0; j_<this->plms[0].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->plms[0].get_W_out(); k_++)
                            cout << img_plms_out[i_*this->plms[0].get_H_out()*this->plms[0].get_W_out() + j_*this->plms[0].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */

                // Resto de capas convolucionales y maxpool ----------------------------
                for(int j=1; j<this->n_capas_conv; ++j)
                {
                    // Capa convolucional 
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j-1];
                    img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[j];
                    img_conv_a = conv_a + img*tam_in_convs + i_conv_in[j];
                    this->convs[j].forwardPropagationGEMM(img_plms_out, img_conv_out, img_conv_a);

                    /*
                    cout << "Output CV: " << this->convs[j].get_n_kernels() << "x" << this->convs[j].get_H_out() << "x" << this->convs[j].get_W_out() << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;
                    */

                    // Capa MaxPool 
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j];
                    img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[j];
                    this->plms[j].forwardPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);

                    /*
                    cout << "Output PM" << endl;
                    for(int i_=0; i_<this->plms[j].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j].get_W_out(); k_++)
                                cout << img_plms_out[i_*this->plms[j].get_H_out()*this->plms[j].get_W_out() + j_*this->plms[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;
                    */

                }  

                // Copiar salida de último MaxPool en flatten
                img_flat_out = flat_outs + img*tam_flat_out; 
                C = this->plms[this->n_capas_conv-1].get_C();
                H_out = this->plms[this->n_capas_conv-1].get_H_out();
                W_out = this->plms[this->n_capas_conv-1].get_W_out();
                //img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[this->n_capas_conv-1];

                for(int i_=0; i_<C; i_++)    
                    for(int j_=0; j_<H_out; j_++)    
                        for(int k_=0; k_<W_out; k_++)
                            img_flat_out[i_*H_out*W_out + j_*W_out + k_] = img_plms_out[i_*H_out*W_out + j_*W_out + k_];
            }
            
            
            
            // ---------------------------------------------------------------------------------------------------------------------------
            // Capa totalmente conectada
            // ---------------------------------------------------------------------------------------------------------------------------

            // Inicializar gradientes de pesos
            for(int i_=0; i_<this->fully->get_n_pesos(); i_++)
                grads_pesos_fully[i_] = 0.0;

            // Inicializar gradientes de sesgos
            for(int i_=0; i_<this->fully->get_n_neuronas(); i_++)
                grads_bias_fully[i_] = 0.0;

            // Realizar propagación hacia delante y hacia detrás en la capa totalmente conectada
            this->fully->train_ptr(img_flat_out, this->train_labels, batch, tam_batches[i], grads_pesos_fully, grads_bias_fully, grad_x_fully, a_ptr, z_ptr, grad_x_fully);
            
            // ----------------------------------------------
            // Pesos de la capa totalmente conectada
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media de los gradientes respecto a cada peso
            for(int i_=0; i_<this->fully->get_n_pesos(); i_++)
                grads_pesos_fully[i_] /= tam_batches[i];

            // ----------------------------------------------
            // Bias o Sesgos de la capa totalmente conectada
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media de los gradientes respecto a cada sesgo
            for(int i_=0; i_<this->fully->get_n_neuronas(); i_++)
                grads_bias_fully[i_] /= tam_batches[i];

            
            // Actualizar parámetros de capas totalmente conectadas 
            this->fully->actualizar_parametros_ptr(grads_pesos_fully, grads_bias_fully);

            this->fully->escalar_pesos_ptr(2);
            
            // ---------------------------------------------------------------------------------------------------------------------------
            // Capas convolucionales, de agrupación y aplanado
            // ---------------------------------------------------------------------------------------------------------------------------

            // ----------------------------------------------
            // ----------------------------------------------
            // BackPropagation ------------------------------
            // ----------------------------------------------
            // ----------------------------------------------

            // Inicializar gradientes a 0
            for(int i_=0; i_<tam_kernels_conv; i_++)
                conv_grads_w[i_] = 0.0;

            for(int i_=0; i_<n_bias_conv; i_++)
                conv_grads_bias[i_] = 0.0;

            // Cálculo de gradientes respecto a cada parámetro 
            for(int img=0; img<tam_batches[i]; ++img)
            {
                // Realizar una copia de la imagen "img" de entrenamiento
                //img_in = this->train_imgs + tam_ini*batch[img];
                for(int i_=0; i_<C_ini; i_++)
                    for(int j_=0; j_<H_ini; j_++)
                        for(int k_=0; k_<W_ini; k_++)
                            this->img_in[i_*H_ini*W_ini + j_*W_ini + k_] = this->train_imgs[i_*H_ini*W_ini + j_*W_ini + k_ + tam_ini*batch[img]]; 
            
                // Última capa, su output no tiene padding
                int i_c=this->n_capas_conv-1;

                // Usar grad_x_fully[img] en vez de plms_outs[img][i_c] en la última capa MaxPool
                img_grad_x_fully = grad_x_fully + img*this->fully->get_capas()[0];

                // Capa MaxPool 
                img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[i_c];
                img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[i_c];
                this->plms[i_c].backPropagationGPU(img_conv_out, img_grad_x_fully, img_plms_in_copy);

                
                // Capa convolucional 
                img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[i_c-1];
                img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[i_c];
                img_conv_a = conv_a + img*tam_in_convs + i_conv_in[i_c];
                img_grad_w_conv = conv_grads_w + i_w[i_c];
                img_grad_b_conv = conv_grads_bias + i_b[i_c];

                
                if(this->n_capas_conv > 1)
                    this->convs[i_c].backPropagationGEMM(img_plms_out, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                else
                    this->convs[i_c].backPropagationGEMM(img_in, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                
                for(int j=this->n_capas_conv-2; j>=1; j--)
                {
                    // Capa MaxPool 
                    img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[j];
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j];
                    img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[j];
                    this->plms[j].backPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);

                    // Capa convolucional 
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j-1];
                    img_conv_a = conv_a + img*tam_in_convs + i_conv_in[j];
                    img_grad_w_conv = conv_grads_w + i_w[j];
                    img_grad_b_conv = conv_grads_bias + i_b[j];
                    this->convs[j].backPropagationGEMM(img_plms_out, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                }
                
                
                if(this->n_capas_conv >1)
                {
                    img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[0];
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[0];
                    img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[0];
                    this->plms[0].backPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);

                    img_conv_a = conv_a + img*tam_in_convs + i_conv_in[0];
                    img_grad_w_conv = conv_grads_w + i_w[0];
                    img_grad_b_conv = conv_grads_bias + i_b[0];
                    this->convs[0].backPropagationGEMM(img_in, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                }
                
            }

            
            // ----------------------------------------------
            // Pesos de las capas convolucionales
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media de los gradientes respecto a cada parámetro
            for(int i_=0; i_<tam_kernels_conv; i_++)
                conv_grads_w[i_] /= tam_batches[i];

            // ----------------------------------------------
            // Bias o Sesgos de las capas convolucionales
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media
            for(int i_=0; i_<n_bias_conv; i_++)
                conv_grads_bias[i_] /= tam_batches[i];

            // Actualizar parámetros --------------------------------------------------------------------

            // Actualizar parámetros de capas convolucionales 
            for(int j=0; j<this->n_capas_conv; ++j)
            {
                img_grad_w_conv = conv_grads_w + i_w[j];
                img_grad_b_conv = conv_grads_bias + i_b[j];
                this->convs[j].actualizar_grads_ptr(img_grad_w_conv, img_grad_b_conv);
            }
            
            // Actualizar parámetros de capas convolucionales 
            for(int j=0; j<this->n_capas_conv; ++j)
                this->convs[j].escalar_pesos_ptr(2);
        }
        
        /*
        fin = high_resolution_clock::now();
        duration = duration_cast<seconds>(fin - ini);
        */
        cout << "Época: " << ep << ",                                           " << duration.count() << " (s)" << endl;
        //cout << "Época: " << ep << ",                                           " << t2-t1 << " (s) " << endl;

        evaluar_modelo();
        
        
    }
    //evaluar_modelo_en_test();
   

    // Liberar memoria
    free(grad_x_fully); free(flat_outs); free(fully_grad_a); free(conv_grads_bias); free(grads_bias_fully); free(grads_pesos_fully); free(convs_outs); free(plms_outs); free(conv_grads_w);
    free(plms_in_copys); free(conv_a); free(indices); free(batch); free(tam_batches);
}

void CNN::mostrar_ptr(float *x, int C, int H, int W)
{
    cout << "\nX\n";
    for(int j=0; j<C; j++)
    {
        for(int k=0; k<H; k++)
        {
            for(int p=0; p<W; p++)
                cout << x[j*H*W + k*W + p] << " ";
            cout << endl;
        }
        cout << endl;
    }
    cout << endl;
}


void CNN::prueba()
{
    int C_in = this->convs[0].get_C(), H_in = this->convs[0].get_H(), W_in = this->convs[0].get_W(),
        C_out = this->convs[0].get_n_kernels(), H_out = this->convs[0].get_H_out(), W_out = this->convs[0].get_W_out();
    float * input = (float *)malloc(C_in*H_in*W_in * sizeof(float)),
          * output = (float *)malloc(C_out*H_out*W_out * sizeof(float)),
          * output_a = (float *)malloc(C_out*H_out*W_out * sizeof(float));
    
    Convolutional conv(this->convs[0].get_n_kernels(), this->convs[0].get_kernel_fils(), this->convs[0].get_kernel_cols(), C_in, H_in, W_in, 0.1);
    
    checkCudaErrors(hipGetLastError());
    cout << "Entro\n";
    this->convs[0].forwardPropagationGEMM(input, output, output_a);
    //conv.forwardPropagationGEMM(input, output, output_a);
    checkCudaErrors(hipGetLastError());
    cout << "Salgo\n";

    free(input); free(output); free(output_a);
}

/*
    @brief  Evalúa el modelo sobre los datos de entrenamiento. Las medidas de evaluación son Accuracy y Entropía Cruzada
*/
void CNN::evaluar_modelo()
{
    float acc=0.0,entr=0.0;
    int C, H, W;

    int j1;
    
    // Realizar la propagación hacia delante
    for(int img=0; img<this->n_imagenes; ++img)
    {
        // Copiar imagen de entrenamiento en img_in
        C = this->convs[0].get_C();
        H = this->convs[0].get_H();
        W = this->convs[0].get_W();
        for(int i=0; i<C; i++)
            for(int j=0; j<H; j++)
                for(int k=0; k<W; k++)
                    img_in[i*H*W + j*W + k] = train_imgs[i*H*W + j*W + k + img*C*H*W];
        
        // Capas convolucionales y maxpool ----------------------------
        for(int i=0; i<this->n_capas_conv; ++i)
        {
            // Capa convolucional 
            this->convs[i].forwardPropagationGEMM(this->img_in, this->img_out, this->conv_a);

            // Copiar img_out en img_in
            C = this->convs[i].get_n_kernels();
            H = this->convs[i].get_H_out();
            W = this->convs[i].get_W_out();

            for(int i=0; i<C; i++)
                for(int j=0; j<H; j++)
                    for(int k=0; k<W; k++)
                        this->img_in[i*H*W + j*W + k] = this->img_out[i*H*W + j*W + k];

            // Capa MaxPool 
            this->plms[i].forwardPropagationGPU(this->img_in, this->img_out, this->img_in_copy);

            // Copiar img_out en img_in
            H = this->plms[i].get_H_out();
            W = this->plms[i].get_W_out();
            for(int i=0; i<C; i++)
                for(int j=0; j<H; j++)
                    for(int k=0; k<W; k++)
                        this->img_in[i*H*W + j*W + k] = this->img_out[i*H*W + j*W + k];
        }
    }
    
    // Cada hebra obtiene el accuracy y la entropía cruzada sobre una porción de imágenes
    acc = (*this->fully).accuracy_ptr(img_out, this->train_labels, n_imagenes, a_ptr, z_ptr);
    entr = (*this->fully).cross_entropy_ptr(img_out, this->train_labels, n_imagenes, a_ptr, z_ptr);

    // Realizar media y obtener valores finales
    acc = acc / n_imagenes * 100;
    entr = -entr / n_imagenes;

    cout << "Accuracy: " << acc << " %,  ";

    cout << "Entropía cruzada: " << entr << ",         " << endl << endl;    
    checkCudaErrors(hipGetLastError());
}

/*
    @brief  Evalúa el modelo sobre los datos de test. Las medidas de evaluación son Accuracy y Entropía Cruzada
*/
void CNN::evaluar_modelo_en_test()
{
    /*
    int n=this->test_imgs.size();
    double t1, t2;
    vector<vector<vector<float>>> img_in, img_out, img_in_copy, conv_a;
    
    vector<float> flat_out; 
    float acc ,entr;

    vector<vector<float>> flat_outs(n);

    // Inicialización de parámetros
    //t1 = omp_get_wtime();
    acc = 0.0;
    entr = 0.0;


    // Popagación hacia delante
    for(int img=0; img<n; img++)
    {
        img_in = this->test_imgs[img];

        // Capas convolucionales y maxpool ----------------------------
        for(int i=0; i<this->n_capas_conv; ++i)
        {
            // Capa convolucional 
            img_out = this->outputs[i*2];
            conv_a = img_out;
            this->convs[i].forwardPropagation(img_in, img_out, conv_a);
            img_in = img_out;

            // Capa MaxPool 
            img_out = this->outputs[i*2+1];
            img_in_copy = img_in;

            int pad_sig = 0;    // Padding de la siguiente capa convolucional
            if(this->n_capas_conv > i+1)
                pad_sig = this->padding[i+1];

            this->plms[i].forwardPropagation(img_in, img_out, img_in_copy, pad_sig);
            img_in = img_out;
        }
        
        // Capa de aplanado
        (*this->flat).forwardPropagation(img_out, flat_out);

        flat_outs[img] = flat_out;
    }
    
    // Cada hebra obtiene el accuracy y la entropía cruzada sobre una porción de imágenes
    acc = (*this->fully).accuracy(flat_outs,this->test_labels);
    entr = (*this->fully).cross_entropy(flat_outs, this->test_labels);

    // Realizar media y obtener valores finales
    acc = acc / n * 100;
    entr = -entr / n;

    //t2 = omp_get_wtime();

    cout << "\n------------- RESULTADOS EN TEST --------------- " << endl;
    cout << "Accuracy: " << acc << " %,  ";


    cout << "Entropía cruzada: " << entr << ",         " << endl << endl;
    //cout << "Entropía cruzada: " << entr << ",         " << t2 - t1 << " (s) " << endl << endl;
    */
}

/*
int main()
{
    //vector<vector<int>> capas_conv = {{3, 3, 3}, {3, 5, 5}}, tams_pool = {{2, 2}, {2, 2}};
    int C=2, H=10, W=10, n_capas_fully = 2, n_capas_conv = 2, n_imagenes = 5, n_clases = 4;
    int *capas_fully = (int *)malloc(n_capas_fully * sizeof(int)),
        *capas_conv = (int *)malloc(n_capas_conv*3 * sizeof(int)),
        *capas_pool = (int *)malloc(n_capas_conv*2 * sizeof(int)),
        *padding = (int *)malloc(n_capas_conv * sizeof(int));
        
    float *X = (float *)malloc(n_imagenes*C*H*W * sizeof(float)),
        *Y = (float *)malloc(n_imagenes*n_clases * sizeof(float));

    float lr = 0.0001;
    int i=0;
    capas_fully[0] = 2;
    capas_fully[1] = n_clases;

    // Primera capa convolucional
    capas_conv[i*3 +0] = 3;      // 4 kernels
    capas_conv[i*3 +1] = 3;      // kernels de 3 filas
    capas_conv[i*3 +2] = 3;      // kernels de 2 columnas

    i = 1;
    // Segunda capa convolucional
    capas_conv[i*3 +0] = 3;      // 7 kernels
    capas_conv[i*3 +1] = 3;      // kernels de 5 filas
    capas_conv[i*3 +2] = 3;      // kernels de 5 columnas

    i=0;
    // Primera capa MaxPool
    capas_pool[i*2 +0] = 2;      // kernels de 2 filas
    capas_pool[i*2 +1] = 2;      // kernels de 2 columnas

    i = 1;
    // Segunda capa MaxPool
    capas_pool[i*2 +0] = 2;      // kernels de 2 filas
    capas_pool[i*2 +1] = 2;      // kernels de 2 columnas
    
    // Padding
    padding[0] = 0;
    padding[1] = 0;

    // Input
    for(int i=0; i<n_imagenes*C*H*W; i++)
        X[i] = i;

    // Etiquetas
    for(int i=0; i<n_imagenes; i++)
        for(int j=0; j<n_clases; j++)
            Y[i*n_clases + j] = 0.0;

    // Poner que todas las imágenes pertecen a la clase 1, por ejemplo
    for(int i=0; i<n_imagenes; i++)
        Y[i*n_clases + 1] = 1.0;

    CNN cnn(capas_conv, n_capas_conv, capas_pool, padding, capas_fully, n_capas_fully, C, H, W, lr);
    //CNN cnn(capas_conv, n_capas_conv, capas_pool, padding, capas_fully, n_capas_fully, C, H-2*padding[0], W-2*padding[0], lr);
    cnn.mostrar_arquitectura();
    cnn.set_train(X, Y, n_imagenes, n_clases, C, H, W);
    //cnn.evaluar_modelo();
    cnn.train(10, 2);

    free(capas_fully); free(capas_conv); free(capas_pool); free(padding);
    return 0;
}
*/