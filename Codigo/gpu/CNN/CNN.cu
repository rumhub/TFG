#include "hip/hip_runtime.h"
#include "CNN.h"

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }else
    {
        cout << "Todo correcto!" << endl;
    }
}


/*
    CONSTRUCTOR de la clase CNN
    --------------------------------------
  
    @capas_conv     Indica el número de capas convolucionales, así como la estructura de cada una. Habrá "capas_conv.size()" capas convolucionales, y la estructura de la capa i 
                    vendrá dada por capas_conv[i]. De esta forma. capas_conv[i] = {3, 2, 2} corresponde a un kernel 3x2x2, por ejemplo.
    @tams_pool      Indica el número de capas de agrupación, así como la estructura de cada una. tams_pool[i] = {2,2} indica un tamaño de ventana de agrupamiento  de 2x2.
    @padding        Indica el nivel de padding de cada capa convolucional. padding[i] corresponde con el nivel de padding a aplicar en la capa capas_conv[i].
    @capas_fully    Vector de enteros que indica el número de neuronas por capa dentro de la capa totalmente conectada. Habrá capas.size() capas y cada una contendrá capas[i] neuronas.
    @input          Volumen 3D de entrada. Se tendrán en cuenta sus dimensiones para crear las estructuras necesarias y permitir un posterior entrenamiento con volúmenes de iguales dimensiones.
    @lr             Learning Rate o Tasa de Aprendizaje
*/
CNN::CNN(int *capas_conv, int n_capas_conv, int *tams_pool, int *padding, int *capas_fully, int n_capas_fully, int C, int H, int W, const float &lr, const int mini_batch)
{
    int * i_capas_conv = nullptr;
    int * i_capas_pool = nullptr;

    // Ejemplo de uso, capas_conv[0] = {16, 3, 3}

    if(C <= 0)
    {
        cout << "ERROR. Hay que proporcionar un input a la red. \n";
        exit(-1);
    }

    this->n_capas_conv = n_capas_conv;
    this->lr = lr;
    this->convs = new Convolutional[this->n_capas_conv];
    this->plms = new PoolingMax[this->n_capas_conv];
    this->padding = (int *)malloc(n_capas_conv * sizeof(int));
    this->n_clases = capas_fully[n_capas_fully-1];
    this->max_H = 0;
    this->max_W = 0;
    this->max_C = 0;
    this->i_conv_out = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_conv_in = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_plm_out = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_plm_in = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_w = (int *)malloc(n_capas_conv * sizeof(int));
    this->i_b = (int *)malloc(n_capas_conv * sizeof(int));

    for(int i=0; i<n_capas_conv; i++)
        this->padding[i] = padding[i];

    // Padding de la primera capa
    H += 2*padding[0];
    W += 2*padding[0];
    
    if(max_C < C)
        max_C = C;

    if(max_H < H)
        max_H = H;
    
    if(max_W < W)
        max_W = W;

    // Inicializar capas convolucionales y maxpool --------------------------------------------
    for(int i=0; i<n_capas_conv; ++i)
    {   
        i_capas_conv = capas_conv + 3*i;
        i_capas_pool = tams_pool + 2*i;

        // Capas convolucionales ------------------------------------------------
        //                  nºkernels          filas_kernel      cols_kernel
        Convolutional conv(i_capas_conv[0], i_capas_conv[1], i_capas_conv[2], C, H, W, lr);
        this->convs[i].copiar(conv);

        // H_out = H - K + 1
        C = i_capas_conv[0];
        H = H - i_capas_conv[1] + 1;
        W = W - i_capas_conv[2] + 1;

        if(max_C < C)
            max_C = C;

        if(max_H < H)
            max_H = H;
        
        if(max_W < W)
            max_W = W;

        // Capas MaxPool -----------------------------------------------------------
        int pad_sig = 0;    // Padding de la siguiente capa convolucional
        if(this->n_capas_conv > i+1)
            pad_sig = this->padding[i+1];
        //           filas_kernel_pool  cols_kernel_pool
        PoolingMax plm(i_capas_pool[0], i_capas_pool[1], C, H, W, pad_sig);
        this->plms[i].copiar(plm);

        // H_out = H / K + 2*pad
        H = H / i_capas_pool[0] + 2*pad_sig;
        W = W / i_capas_pool[0] + 2*pad_sig;

        if(max_C < C)
            max_C = C;

        if(max_H < H)
            max_H = H;
        
        if(max_W < W)
            max_W = W;
    }

    
    // Inicializar capa fullyconnected -----------------------------------------
    int *capas_fully_ptr = (int *)malloc((n_capas_fully+1) * sizeof(int));

    capas_fully_ptr[0] = C*H*W;

    for(int i=1; i<n_capas_fully+1; i++)
        capas_fully_ptr[i] = capas_fully[i-1];


    this->fully = new FullyConnected(capas_fully_ptr, n_capas_fully+1, lr, mini_batch);

    // Reserva de espacio para posteriores operaciones
    int tam_img_max = max_C*max_H*max_W;
    //int tam_img_max = max_C*max_H*max_W;

    this->img_in = (float *)malloc(tam_img_max * sizeof(float));
    this->img_out = (float *)malloc(tam_img_max * sizeof(float)); 
    this->img_in_copy = (float *)malloc(tam_img_max * sizeof(float)); 
    this->conv_a = (float *)malloc(tam_img_max * sizeof(float));
    this->a_ptr = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float));
    this->grad_a_ptr = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float));
    this->z_ptr = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float)); 

    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------
    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------
    for(int i=0; i<tam_img_max; i++)
    {
        this->img_in[i] = 1.0;
        this->img_out[i] = 2.0;
        this->conv_a[i] = 3.0;
    }
    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------
    // Borrar ----------------------------------------------------------------------------------------------------------------------------------------------------

    int i_out_c = 0, i_in_c = 0, i_out_p = 0, i_in_p = 0, i_w_ = 0, i_b_ = 0;
    for(int i=0; i<n_capas_conv; i++)
    {
        // Convolucional
        this->i_conv_in[i] = i_in_c;
        i_in_c += this->convs[i].get_C() * this->convs[i].get_H() * this->convs[i].get_W(); 

        this->i_conv_out[i] = i_out_c;
        i_out_c += this->convs[i].get_n_kernels() * this->convs[i].get_H_out() * this->convs[i].get_W_out();   

        // Agrupación máxima
        this->i_plm_in[i] = i_in_p;
        i_in_p += this->plms[i].get_C() * this->plms[i].get_H() * this->plms[i].get_W(); 

        this->i_plm_out[i] = i_out_p;
        i_out_p += this->plms[i].get_C() * this->plms[i].get_H_out() * this->plms[i].get_W_out();  

        i_w[i] = i_w_;
        i_w_ += this->convs[i].get_n_kernels() * this->convs[i].get_C() * this->convs[i].get_kernel_fils() * this->convs[i].get_kernel_cols();

        i_b[i] = i_b_;
        i_b_ += this->convs[i].get_n_kernels();
    }

    // Liberar memoria
    free(capas_fully_ptr);
}

/*
    @brief  Muestra la arquitectura de la red
*/
void CNN::mostrar_arquitectura()
{
    cout << "\n-----------Arquitectura de la red-----------\n";
    cout << "Padding por capa: ";
    for(int i=0; i<this->n_capas_conv-1; i++)
        cout << this->padding[i] << ", ";
    cout << this->padding[this->n_capas_conv-1];
    cout << endl;
    
    for(int i=0; i<this->n_capas_conv; i++)
    {
        cout << "Dimensiones de entrada a " << this->convs[i].get_n_kernels() << " kernels " << this->convs[i].get_kernel_fils() << "x" << this->convs[i].get_kernel_cols() << " convolucionales: " << this->convs[i].get_C() << "x" << this->convs[i].get_H() << "x" << this->convs[i].get_W() << endl;
        cout << "Dimensiones de entrada a un kernel " << this->plms[i].get_kernel_fils() << "x" << this->plms[i].get_kernel_cols() << " MaxPool: " << this->plms[i].get_C() << "x" << this->plms[i].get_H() << "x" << this->plms[i].get_W() << endl;
    }

    // Volúmen de salida de la última capa MaxPool
    cout << "Dimensiones de salida de un kernel " << this->plms[this->n_capas_conv-1].get_kernel_fils() << "x" << this->plms[this->n_capas_conv-1].get_kernel_cols() << " MaxPool: " << this->plms[this->n_capas_conv-1].get_C() << "x" << this->plms[this->n_capas_conv-1].get_H_out() << "x" << this->plms[this->n_capas_conv-1].get_W_out() << endl;

    // Capas totalmente conectadas
    int * capas = this->fully->get_capas();

    cout << "Capas totalmente concetadas: ";
    for(int i=0; i<this->fully->get_n_capas()-1; i++)
        cout << capas[i] << ", ";
    cout << capas[this->fully->get_n_capas()-1];
    
    cout << endl;
}

void CNN::mostrar_train_img(int n_img)
{
    int C = this->convs[0].get_C(),
        H = this->convs[0].get_H(),
        W = this->convs[0].get_W();
    float *img_train = this->train_imgs + n_img*C*H*W;

    cout << "\nImagen " << n_img << endl;
    for(int j=0; j<C; j++)
    {
        for(int k=0; k<H; k++)
        {
            for(int p=0; p<W; p++)
                cout << img_train[j*H*W + k*W + p] << " ";
            cout << endl;
        }
        cout << endl;
    }
    cout << endl;
}


void CNN::set_train(float *x, float *y, int n_imgs, int n_clases, int C, int H, int W)
{
    n_imgs -= 1;
    H += 2*this->padding[0];
    W += 2*this->padding[0];
    this->n_imagenes = n_imgs * n_clases;
    this->train_imgs = (float *)malloc(n_imagenes*C*H*W * sizeof(float));
    this->train_labels = (float *)malloc(n_imagenes*n_clases * sizeof(float));

    int tam_flat_out = this->plms[this->n_capas_conv-1].get_C() * this->plms[this->n_capas_conv-1].get_H_out() * this->plms[this->n_capas_conv-1].get_W_out();
    this->flat_outs = (float *)malloc(this->n_imagenes* tam_flat_out * sizeof(float));
    if(this->n_clases != n_clases)
        cout << "\n\nError. Número de clases distinto al establecido previamente en la arquitectura de la red. " << this->n_clases << " != " << n_clases << endl << endl;

    for(int i=0; i<n_imagenes*C*H*W; i++)
        train_imgs[i] = x[i];
    
    for(int i=0; i<n_imagenes*n_clases; i++)
        train_labels[i] = y[i];

    /*
    // Mostrar imágenes
    cout << "\nX\n";
    for(int i=0; i<this->n_imagenes; i++)
    {
        for(int j=0; j<C; j++)
        {
            for(int k=0; k<H; k++)
            {
                for(int p=0; p<W; p++)
                    cout << x[i*C*H*W + j*H*W + k*W + p] << " ";
                cout << endl;
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << endl;

    cout << "\nY\n";
    for(int i=0; i<this->n_imagenes; i++)
    {
        for(int j=0; j<n_clases; j++)
            cout << y[i*n_clases + j] << " ";
        cout << endl;
    }
    cout << endl;
    */
}

/*
    @brief  Aplica padding sobre una imagen sin aumentar su tamaño
    @input  Imagen sobre la cual aplicar padding
    @pad    Nivel de padding a aplicar
    @return Imagen @input con padding interno aplicado
*/
void CNN::padding_interno_ptr(float *input, int C, int H, int W, const int &pad)
{
    for(int i=0; i<C; ++i)
    {
        // Primeras "pad" filas se igualan a 0.0
        for(int j=0; j<pad; ++j)
            for(int k=0; k<H; ++k)
            input[i*H*W + j*W + k] = 0.0; 

        // Últimas "pad" filas se igualan a 0.0
        for(int j=H-1; j>=H - pad; j--)
            for(int k=0; k<H; ++k)
            input[i*H*W + j*W + k] = 0.0; 
        
        // Por cada fila
        for(int k=0; k<H; ++k)
        {
            // Primeras "pad" casillas se igualan a 0.0
            for(int j=0; j<pad; ++j)
                input[i*H*W + j*W + k] = 0.0;

            // Últimas "pad" casillas se igualan a 0.0
            for(int j=W-1; j>=W - pad; j--)
                input[i*H*W + j*W + k] = 0.0;
        }
    }    
}

/*
    @brief  Aplica padding sobre una imagen sin aumentar su tamaño
    @input  Imagen sobre la cual aplicar padding
    @pad    Nivel de padding a aplicar
    @return Imagen @input con padding interno aplicado
*/
void CNN::padding_interno(vector<vector<vector<float>>> &input, const int &pad)
{
    for(int i=0; i<input.size(); ++i)
    {
        // Primeras "pad" filas se igualan a 0.0
        for(int j=0; j<pad; ++j)
            for(int k=0; k<input[i].size(); ++k)
            input[i][j][k] = 0.0; 

        // Últimas "pad" filas se igualan a 0.0
        for(int j=input[i].size()-1; j>=input[i].size() - pad; j--)
            for(int k=0; k<input[i].size(); ++k)
            input[i][j][k] = 0.0; 
        
        // Por cada fila
        for(int k=0; k<input[i].size(); ++k)
        {
            // Primeras "pad" casillas se igualan a 0.0
            for(int j=0; j<pad; ++j)
                input[i][k][j] = 0.0;

            // Últimas "pad" casillas se igualan a 0.0
            for(int j=input[i][k].size()-1; j>=input[i][k].size() - pad; j--)
                input[i][k][j] = 0.0;
        }
    }    
}


void shuffle(int *vec, int tam_vec, mt19937& rng) {
    for (int i = tam_vec - 1; i > 0; --i) {
        std::uniform_int_distribution<int> dist(0, i);
        int j = dist(rng);
        std::swap(vec[i], vec[j]);
    }
}


/*
                cout << "Input" << endl;
                for(int i=0; i<this->convs[0].get_C(); i++)
                {
                    for(int j=0; j<this->convs[0].get_H(); j++)
                    {
                        for(int k=0; k<this->convs[0].get_W(); k++)
                            cout << img_train[i*this->convs[0].get_H()*this->convs[0].get_W() + j*this->convs[0].get_W() + k] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                int pepe;
                cin >> pepe;

                cout << "Output" << endl;
                for(int i=0; i<this->convs[0].get_n_kernels(); i++)
                {
                    for(int j=0; j<this->convs[0].get_H_out(); j++)
                    {
                        for(int k=0; k<this->convs[0].get_W_out(); k++)
                            cout << img_conv_out[i*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j*this->convs[0].get_W_out() + k] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                cin >> pepe;
*/


void CNN::train(int epocas, int mini_batch)
{
    
    auto ini = high_resolution_clock::now();
    auto fin = high_resolution_clock::now();
    auto duration = duration_cast<seconds>(fin - ini);

    int n=this->n_imagenes;
    int C, H_out, W_out;
   
    int tam_in_convs = 0, tam_out_convs = 0, tam_in_pools = 0, tam_out_pools = 0, tam_kernels_conv = 0, 
        tam_flat_out = this->plms[this->n_capas_conv-1].get_C() * this->plms[this->n_capas_conv-1].get_H_out() * this->plms[this->n_capas_conv-1].get_W_out(),
        n_bias_conv = 0;

    for(int i=0; i<this->n_capas_conv; i++)
    {
        tam_kernels_conv += this->convs[i].get_n_kernels() * this->convs[i].get_C() * this->convs[i].get_kernel_fils() * this->convs[i].get_kernel_cols(); 
        tam_in_convs += this->convs[i].get_C() * this->convs[i].get_H() * this->convs[i].get_W(); 
        tam_out_convs += this->convs[i].get_n_kernels() * this->convs[i].get_H_out() * this->convs[i].get_W_out(); 
        tam_out_pools += this->plms[i].get_C() * this->plms[i].get_H_out() * this->plms[i].get_W_out(); 
        tam_in_pools += this->plms[i].get_C() * this->plms[i].get_H() * this->plms[i].get_W(); 
        n_bias_conv += this->convs[i].get_n_kernels();
    }


    float *grad_x_fully = (float *)malloc(mini_batch* this->fully->get_capas()[0] * sizeof(float)), // Capa totalmente conectada
          *grads_bias_fully = (float *)malloc(this->fully->get_n_neuronas() * sizeof(float)),
          *grads_pesos_fully = (float *)malloc(this->fully->get_n_pesos() * sizeof(float)),
          *flat_outs_batch = (float *)malloc(mini_batch* tam_flat_out * sizeof(float)),                   // Capa de aplanado
          *plms_outs = (float *)malloc(mini_batch * tam_out_pools * sizeof(float)),                 // Capa de agrupación máxima
          *plms_in_copys = (float *)malloc(mini_batch * tam_in_pools* sizeof(float)), 
          *conv_grads_w = (float *)malloc(tam_kernels_conv * sizeof(float)),                        // Capa convolucional
          *conv_grads_bias = (float *)malloc(n_bias_conv * sizeof(float)),
          *convs_outs = (float *)malloc(mini_batch * tam_out_convs * sizeof(float)), 
          *conv_a = (float *)malloc(mini_batch * tam_out_convs * sizeof(float)); 

    float *img_train = nullptr;
    float *img_conv_out = nullptr;
    float *img_conv_a = nullptr;
    float *img_plms_out = nullptr;
    float *img_plms_in_copy = nullptr;
    float *img_flat_out = nullptr;
    float *img_grad_x_fully = nullptr;
    float *img_grad_w_conv = nullptr;
    float *img_grad_b_conv = nullptr;


    float *y_batch = (float *)malloc(mini_batch*n_clases * sizeof(float));

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------
    /*
    int cont = 0, cont_b = 0;
    for(int i=0; i<this->n_capas_conv; i++)
    {
        int n_K = this->convs[i].get_n_kernels(), C = this->convs[i].get_C(), H = this->convs[i].get_kernel_fils(), W = this->convs[i].get_kernel_cols();
        img_grad_w_conv = conv_grads_w + i_w[i];
        img_grad_b_conv = conv_grads_bias + i_b[i];
        
        for(int n=0; n<n_K; n++)
        {
            // Bias
            img_grad_b_conv[n] = cont_b++;

            // Conv
            for(int j=0; j<C; j++)
                for(int k=0; k<H; k++)
                    for(int p=0; p<W; p++)
                        img_grad_w_conv[n*C*H*W + j*H*W + k*W + p] = cont++;
        }

    }

    // Mostrar gradientes de pesos
    for(int i=0; i<this->n_capas_conv; i++)
    {
        int n_K = this->convs[i].get_n_kernels(), C = this->convs[i].get_C(), H = this->convs[i].get_kernel_fils(), W = this->convs[i].get_kernel_cols();
        img_grad_w_conv = conv_grads_w + i_w[i];
        img_grad_b_conv = conv_grads_bias + i_b[i];
        
        cout << "Capa conv " << i <<  ", " << n_K << "x" << C << "x" << H << "x" << W << endl;
        for(int n=0; n<n_K; n++)
        {
            cout << "(" << i << ", " << n << ") Bias: " << img_grad_b_conv[n] << endl;
            for(int j=0; j<C; j++)
            {
                for(int k=0; k<H; k++)
                {
                    for(int p=0; p<W; p++)
                        cout << img_grad_w_conv[n*C*H*W + j*H*W + k*W + p] << " ";
                    cout << endl;
                }
                cout << endl;
            }
            cout << endl;
        }
    }
    */


    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------


    const int M = n / mini_batch;
    int pad_sig, C_ini = this->convs[0].get_C(), H_ini = this->convs[0].get_H(), W_ini = this->convs[0].get_W(), tam_ini = C_ini*H_ini*W_ini;

    std::random_device rd;
    std::mt19937 g(rd());

    int n_batches = M;
    if(n % mini_batch != 0)
        n_batches++;
    int *indices = (int *)malloc(n * sizeof(int)),
        *batch = (int *)malloc(mini_batch * sizeof(int)),
        *tam_batches = (int *)malloc(n_batches * sizeof(int));
    
    //-------------------------------------------------
    // Inicializar índices
    //-------------------------------------------------
    // Inicializar vector de índices
    for(int i=0; i<n; ++i)
        indices[i] = i;

    // Inicializar tamaño de mini-batches
    for(int i=0; i<M; ++i)
        tam_batches[i] = mini_batch;
    
    // Último batch puede tener distinto tamaño al resto
    if(n % mini_batch != 0)
        tam_batches[n_batches-1] = n % mini_batch;   

    
    int k1;
    for(int ep=0; ep<epocas; ++ep)
    {
        
        ini = high_resolution_clock::now();

        // Desordenar vector de índices
        shuffle(indices, n, g);

        

    // ---------------------------------------------------------------
    // ---------------------------------------------------------------
    for(int i=0; i<mini_batch; i++)
        for(int j=0; j<n_clases; j++)
            y_batch[i*n_clases + j] = train_labels[indices[i]*n_clases + j];

    cout << "LABELS" << endl;
    for(int i=0; i<mini_batch; i++)
    {
        for(int j=0; j<n_clases; j++)
            cout << train_labels[indices[i]*n_clases + j] << " ";
        cout << endl;
    }
    cout << endl;

    cout << "Y" << endl;
    for(int i=0; i<mini_batch; i++)
    {
        for(int j=0; j<n_clases; j++)
            cout << y_batch[i*n_clases + j] << " ";
        cout << endl;
    }
    cout << endl;

    int k2;
    cin >> k2;
    // ---------------------------------------------------------------
    // ---------------------------------------------------------------




        // ForwardPropagation de cada batch -----------------------------------------------------------------------
        for(int i=0; i<n_batches; ++i)
        {
            
            // Crear el batch para cada hebra ----------------------
            for(int j=0; j<tam_batches[i]; j++)
                batch[j] = indices[mini_batch*i + j];   

            
            for(int img=0; img<tam_batches[i]; ++img)
                for(int j=0; j<this->n_capas_conv; ++j)
                {
                    pad_sig = 0;    // Padding de la siguiente capa convolucional
                    if(this->n_capas_conv > j+1)
                        pad_sig = this->padding[j+1];

                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j];
                    padding_interno_ptr(img_plms_out, this->plms[j].get_C(), this->plms[j].get_H_out(), this->plms[j].get_W_out(), pad_sig);
                }
            
            
            // ---------------------------------------------------------------------------------------
            for(int img=0; img<tam_batches[i]; ++img)
            {
                
                // Primera capa convolucional y maxpool -----------------------
                img_train = this->train_imgs + tam_ini*batch[img];
                img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[0];
                img_conv_a = conv_a + img*tam_out_convs + i_conv_out[0];
                this->convs[0].forwardPropagationGEMM(img_train, img_conv_out, img_conv_a);

                /*
                cout << "Input" << endl;
                for(int i_=0; i_<this->convs[0].get_C(); i_++)
                {
                    for(int j_=0; j_<this->convs[0].get_H(); j_++)
                    {
                        for(int k_=0; k_<this->convs[0].get_W(); k_++)
                            cout << img_train[i_*this->convs[0].get_H()*this->convs[0].get_W() + j_*this->convs[0].get_W() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;

                cout << "Input de verdad " << endl;
                mostrar_train_img(img);

                //cin >> k1;
                
                cout << "Conv A 0 " << endl;
                for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                            cout << img_conv_a[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;

                cout << "Output 0 " << endl;
                for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                            cout << img_conv_out[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */
                
                img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[0];
                img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[0];
                this->plms[0].forwardPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);
                
                /*
                cout << "Input MaxPool 0" << endl;
                for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                            cout << img_conv_out[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;


                cout << "PLM_in_copys 0" << endl;
                for(int i_=0; i_<this->plms[0].get_C(); i_++)
                {
                    for(int j_=0; j_<this->plms[0].get_H(); j_++)
                    {
                        for(int k_=0; k_<this->plms[0].get_W(); k_++)
                            cout << img_plms_in_copy[i_*this->plms[0].get_H()*this->plms[0].get_W() + j_*this->plms[0].get_W() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;

                
                cout << "Output MaxPool 0" << endl;
                for(int i_=0; i_<this->plms[0].get_C(); i_++)
                {
                    for(int j_=0; j_<this->plms[0].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->plms[0].get_W_out(); k_++)
                            cout << img_plms_out[i_*this->plms[0].get_H_out()*this->plms[0].get_W_out() + j_*this->plms[0].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */
                
                
                // Resto de capas convolucionales y maxpool ----------------------------
                for(int j=1; j<this->n_capas_conv; ++j)
                {
                    
                    // Capa convolucional 
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j-1];
                    img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[j];
                    img_conv_a = conv_a + img*tam_out_convs + i_conv_out[j];
                    this->convs[j].forwardPropagationGEMM(img_plms_out, img_conv_out, img_conv_a);

                    /*
                    cout << "Input Conv: " << j << ", " << this->plms[j-1].get_C() << "x" << this->plms[j-1].get_H_out() << "x" << this->plms[j-1].get_W_out() << endl;
                    for(int i_=0; i_<this->plms[j-1].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j-1].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j-1].get_W_out(); k_++)
                                cout << img_plms_out[i_*this->plms[j-1].get_H_out()*this->plms[j-1].get_W_out() + j_*this->plms[j-1].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;

                    cout << "Conv A: " << j << ", " << this->convs[j].get_n_kernels() << "x" << this->convs[j].get_H_out() << "x" << this->convs[j].get_W_out() << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_a[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;

                    cout << "Output Conv: " << j << ", " << this->convs[j].get_n_kernels() << "x" << this->convs[j].get_H_out() << "x" << this->convs[j].get_W_out() << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;
                    */
                    
                    
                    // Capa MaxPool 
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j];
                    img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[j];
                    this->plms[j].forwardPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);
                    
                    /*
                    cout << "Input MaxPool" << j << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;

                    cout << "PLM_in_copys " << j << endl;
                    for(int i_=0; i_<this->plms[j].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j].get_H(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j].get_W(); k_++)
                                cout << img_plms_in_copy[i_*this->plms[j].get_H()*this->plms[j].get_W() + j_*this->plms[j].get_W() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;

                    cout << "Output MaxPool " << j << endl;
                    for(int i_=0; i_<this->plms[j].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j].get_W_out(); k_++)
                                cout << img_plms_out[i_*this->plms[j].get_H_out()*this->plms[j].get_W_out() + j_*this->plms[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;
                    */
                }  

                
                // Copiar salida de último MaxPool en flatten
                img_flat_out = flat_outs_batch + img*tam_flat_out; 
                C = this->plms[this->n_capas_conv-1].get_C();
                H_out = this->plms[this->n_capas_conv-1].get_H_out();
                W_out = this->plms[this->n_capas_conv-1].get_W_out();
                //img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[this->n_capas_conv-1];

                
                for(int i_=0; i_<C; i_++)    
                    for(int j_=0; j_<H_out; j_++)    
                        for(int k_=0; k_<W_out; k_++)
                            img_flat_out[i_*H_out*W_out + j_*W_out + k_] = img_plms_out[i_*H_out*W_out + j_*W_out + k_];
                
                /*
                cout << "Input Flatten (img: " << img << ") " << endl;
                for(int i_=0; i_<C; i_++)
                {
                    for(int j_=0; j_<H_out; j_++)
                    {
                        for(int k_=0; k_<W_out; k_++)
                            cout << img_flat_out[i_*H_out*W_out + j_*W_out + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;


                cout << "Outputput Flatten (img: " << img << ") " << endl;
                for(int i_=0; i_<tam_flat_out; i_++)
                {
                    cout << img_flat_out[i_] << " ";
                }
                cout << endl;
                cout << endl;

                //cin >> k1;
                */
            }

            /*
            cout << "Output Flatten Total: " << endl;
            for(int i_=0; i_<tam_batches[i]; i_++)
            {
                for(int j_=0; j_<tam_flat_out; j_++)
                    cout << flat_outs_batch[i_*tam_flat_out + j_] << " ";
                cout << endl;
            }
            cout << endl;
            cout << endl;

           //cin >> k1;
           */
           

            
            
            
            // ---------------------------------------------------------------------------------------------------------------------------
            // Capa totalmente conectada
            // ---------------------------------------------------------------------------------------------------------------------------

            // Inicializar gradientes de pesos
            for(int i_=0; i_<this->fully->get_n_pesos(); i_++)
                grads_pesos_fully[i_] = 0.0;
            
            // Inicializar gradientes de sesgos
            for(int i_=0; i_<this->fully->get_n_neuronas(); i_++)
                grads_bias_fully[i_] = 0.0;
            
            // Realizar propagación hacia delante y hacia detrás en la capa totalmente conectada
            this->fully->train_ptr(flat_outs_batch, this->train_labels, batch, tam_batches[i], grads_pesos_fully, grads_bias_fully, grad_x_fully, a_ptr, z_ptr, grad_a_ptr);
            
            /*
            cout << "Output Flatten Total: " << endl;
            for(int i_=0; i_<tam_flat_out*tam_batches[i]; i_++)
            {
                cout << flat_outs_batch[i_] << " ";
            }
            cout << endl;
            cout << endl;

            //cin >> k1;
            */
            int n_clases = this->fully->get_capas()[ this->fully->get_n_capas() - 1];
            
            /*
            cout << "Train labels: " << endl;
            for(int i_=0; i_<n_imagenes; i_++)
            {
                for(int j_=0; j_<n_clases; j_++)
                    cout << train_labels[i_*n_clases + j_] << " ";
                cout << endl;
            }
            cout << endl;
            //cin >> k1;

            cout << "Batch: " << endl;
            for(int i_=0; i_<tam_batches[i]; i_++)
                cout << batch[i_] << " ";
            cout << endl;
            cout << endl;  
            //cin >> k1;
            
            cout << "Nº imágenes batch: " << tam_batches[i] << endl;

            cout << "Pesos fully: " << endl;
            */
            float *w_fully = this->fully->get_pesos_ptr();

            int n_capas_fully = this->fully->get_n_capas(),
                * capas_fully = this->fully->get_capas(),
                *i_w_ptr = this->fully->get_i_w_ptr(),
                *i_capa_fully = this->fully->get_i_capa();

            /*
            // Mostrar pesos
            cout << "Pesos fully" << endl;
            for(int i=0; i<n_capas_fully-1; i++)
            {
                cout << "(" << capas_fully[i] << "x" << capas_fully[i+1] << ") " << endl;
                
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)   // Por cada neurona de la capa actual
                {
                    for(int k=0; k<capas_fully[i+1]; k++)     // Por cada neurona de la siguiente capa
                        cout << w_fully[i_w_ptr[i] + j*capas_fully[i+1] + k] << " ";
                    
                    cout << endl;
                }
                cout << endl;
                
            }
            cout << endl;

           //cin >> k1;
            
            cout << "Grad pesos" << endl;
            for(int i=0; i<n_capas_fully-1; i++)
            {
                cout << "(" << capas_fully[i] << "x" << capas_fully[i+1] << ") " << endl;
                
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)   // Por cada neurona de la capa actual
                {
                    for(int k=0; k<capas_fully[i+1]; k++)     // Por cada neurona de la siguiente capa
                        cout << grads_pesos_fully[i_w_ptr[i] + j*capas_fully[i+1] + k] << " ";
                    
                    cout << endl;
                }
                cout << endl;
                
            }
            cout << endl;

           //cin >> k1;

            /*
            // Mostrar neuronas
            cout << "Grad Bias" << endl;
            for(int i=0; i<n_capas_fully; i++)
            {
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)
                    cout << grads_bias_fully[i_capa_fully[i] + j] << " ";
                cout << endl;
                cout << endl;
            }
            cout << endl;

            //cin >> k1;

            
            cout << "Grad X Fully" << endl;
            cout << tam_batches[i] <<  "x" << this->fully->get_capas()[0] << endl << endl;
            for(int i_=0; i_<tam_batches[i]; i_++)
            {
                cout << "Capa " << i_ << endl;
                for(int j=0; j<this->fully->get_capas()[0]; j++)
                    cout << grad_x_fully[i_*this->fully->get_capas()[0] + j] << " ";
                cout << endl;
            }
            cout << endl;

            //cin >> k1;
            */
            
            // ----------------------------------------------
            // Pesos de la capa totalmente conectada
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media de los gradientes respecto a cada peso
            for(int i_=0; i_<this->fully->get_n_pesos(); i_++)
                grads_pesos_fully[i_] /= tam_batches[i];
            
            /*
            cout << "Grad pesos (después media): " << endl;
            for(int i=0; i<n_capas_fully-1; i++)
            {
                cout << "(" << capas_fully[i] << "x" << capas_fully[i+1] << ") " << endl;
                
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)   // Por cada neurona de la capa actual
                {
                    for(int k=0; k<capas_fully[i+1]; k++)     // Por cada neurona de la siguiente capa
                        cout << grads_pesos_fully[i_w_ptr[i] + j*capas_fully[i+1] + k] << " ";
                    
                    cout << endl;
                }
                cout << endl;
                
            }
            cout << endl;

           //cin >> k1;
           */

            // ----------------------------------------------
            // Bias o Sesgos de la capa totalmente conectada
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media de los gradientes respecto a cada sesgo
            for(int i_=0; i_<this->fully->get_n_neuronas(); i_++)
                grads_bias_fully[i_] /= tam_batches[i];

            /*
            cout << "Grad Bias (después media)" << endl;
            for(int i=0; i<n_capas_fully; i++)
            {
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)
                    cout << grads_bias_fully[i_capa_fully[i] + j] << " ";
                cout << endl;
                cout << endl;
            }
            cout << endl;

           //cin >> k1;
           */


                        
            // Actualizar parámetros de capas totalmente conectadas 
            this->fully->actualizar_parametros_ptr(grads_pesos_fully, grads_bias_fully);
            this->fully->escalar_pesos_ptr(2);
            
            w_fully = this->fully->get_pesos_ptr();
            
            /*
            cout << "Pesos fully (después actualización y escalado):" << endl;
            for(int i=0; i<n_capas_fully-1; i++)
            {
                cout << "(" << capas_fully[i] << "x" << capas_fully[i+1] << ") " << endl;
                
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)   // Por cada neurona de la capa actual
                {
                    for(int k=0; k<capas_fully[i+1]; k++)     // Por cada neurona de la siguiente capa
                        cout << w_fully[i_w_ptr[i] + j*capas_fully[i+1] + k] << " ";
                    
                    cout << endl;
                }
                cout << endl;
                
            }
            cout << endl;

            //cin >> k1;
            

            float *bias_fully = this->fully->get_bias_ptr();
            /*
            cout << "Bias (después actualización y escalado):" << endl;
            for(int i=0; i<n_capas_fully; i++)
            {
                cout << "Capa " << i << endl;
                for(int j=0; j<capas_fully[i]; j++)
                    cout << bias_fully[i_capa_fully[i] + j] << " ";
                cout << endl;
                cout << endl;
            }
            cout << endl;

            //cin >> k1;
            */
            
            
            // ---------------------------------------------------------------------------------------------------------------------------
            // Capas convolucionales, de agrupación y aplanado
            // ---------------------------------------------------------------------------------------------------------------------------

            // ----------------------------------------------
            // ----------------------------------------------
            // BackPropagation ------------------------------
            // ----------------------------------------------
            // ----------------------------------------------

            //cout << " ----------- BACKPROP ----------- " << endl;
            // Inicializar gradientes a 0
            for(int i_=0; i_<tam_kernels_conv; i_++)
                conv_grads_w[i_] = 0.0;
            
            for(int i_=0; i_<n_bias_conv; i_++)
                conv_grads_bias[i_] = 0.0;
            
            // Cálculo de gradientes respecto a cada parámetro 
            for(int img=0; img<tam_batches[i]; ++img)
            {
                
                // Realizar una copia de la imagen "img" de entrenamiento
                //img_in = this->train_imgs + tam_ini*batch[img];
                for(int i_=0; i_<C_ini; i_++)
                    for(int j_=0; j_<H_ini; j_++)
                        for(int k_=0; k_<W_ini; k_++)
                            this->img_in[i_*H_ini*W_ini + j_*W_ini + k_] = this->train_imgs[i_*H_ini*W_ini + j_*W_ini + k_ + tam_ini*batch[img]]; 
                
                // Última capa, su output no tiene padding
                int i_c=this->n_capas_conv-1;
                
                // Usar grad_x_fully[img] en vez de plms_outs[img][i_c] en la última capa MaxPool
                img_grad_x_fully = grad_x_fully + img*this->fully->get_capas()[0];
                

                // -------------------------------------------------------------------------------------
                // -------------------------------------------------------------------------------------
                
                C = this->plms[this->n_capas_conv-1].get_C();
                H_out = this->plms[this->n_capas_conv-1].get_H_out();
                W_out = this->plms[this->n_capas_conv-1].get_W_out();
                /*
                cout << "Input Flatten (img: " << img << ") " << endl;
                for(int i_=0; i_<C; i_++)
                {
                    for(int j_=0; j_<H_out; j_++)
                    {
                        for(int k_=0; k_<W_out; k_++)
                            cout << img_grad_x_fully[i_*H_out*W_out + j_*W_out + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */

                // -------------------------------------------------------------------------------------
                // -------------------------------------------------------------------------------------
                                
                // Capa MaxPool 
                img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[i_c];
                img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[i_c];
                this->plms[i_c].backPropagationGPU(img_conv_out, img_grad_x_fully, img_plms_in_copy);

                /*
                cout << "Input MaxPool b: " << i_c << endl;
                for(int i_=0; i_<this->convs[i_c].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[i_c].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[i_c].get_W_out(); k_++)
                            cout << img_conv_out[i_*this->convs[i_c].get_H_out()*this->convs[i_c].get_W_out() + j_*this->convs[i_c].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;

                cout << "PLM_in_copys b: " << i_c << endl;
                for(int i_=0; i_<this->plms[i_c].get_C(); i_++)
                {
                    for(int j_=0; j_<this->plms[i_c].get_H(); j_++)
                    {
                        for(int k_=0; k_<this->plms[i_c].get_W(); k_++)
                            cout << img_plms_in_copy[i_*this->plms[i_c].get_H()*this->plms[i_c].get_W() + j_*this->plms[i_c].get_W() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */


                
                // Capa convolucional 
                img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[i_c-1];
                img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[i_c];
                img_conv_a = conv_a + img*tam_out_convs + i_conv_out[i_c];
                img_grad_w_conv = conv_grads_w + i_w[i_c];
                img_grad_b_conv = conv_grads_bias + i_b[i_c];

                
                if(this->n_capas_conv > 1)
                    this->convs[i_c].backPropagationGEMM(img_plms_out, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                else
                    this->convs[i_c].backPropagationGEMM(img_in, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                
                /*
                cout << "Output MaxPool " << i_c-1 << endl;
                for(int i_=0; i_<this->plms[i_c-1].get_C(); i_++)
                {
                    for(int j_=0; j_<this->plms[i_c-1].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->plms[i_c-1].get_W_out(); k_++)
                            cout << img_plms_out[i_*this->plms[i_c-1].get_H_out()*this->plms[i_c-1].get_W_out() + j_*this->plms[i_c-1].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;

                cout << "Conv A: " << i_c << ", " << this->convs[i_c].get_n_kernels() << "x" << this->convs[i_c].get_H_out() << "x" << this->convs[i_c].get_W_out() << endl;
                for(int i_=0; i_<this->convs[i_c].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[i_c].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[i_c].get_W_out(); k_++)
                            cout << img_conv_a[i_*this->convs[i_c].get_H_out()*this->convs[i_c].get_W_out() + j_*this->convs[i_c].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;

                cout << "Conv Out b: " << i_c << endl;
                for(int i_=0; i_<this->convs[i_c].get_n_kernels(); i_++)
                {
                    for(int j_=0; j_<this->convs[i_c].get_H_out(); j_++)
                    {
                        for(int k_=0; k_<this->convs[i_c].get_W_out(); k_++)
                            cout << img_conv_out[i_*this->convs[i_c].get_H_out()*this->convs[i_c].get_W_out() + j_*this->convs[i_c].get_W_out() + k_] << " ";
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

                //cin >> k1;
                */
                //cout << "Grad Pesos Conv b: " << i_c << endl;
                int n_kernels = this->convs[i_c].get_n_kernels(), K = this->convs[i_c].get_kernel_fils();
                C = this->convs[i_c].get_C();
                /*
                for(int i=0; i<n_kernels; ++i)
                {
                    for(int j=0; j<C; ++j)
                    {
                        for(int k=0; k<K; ++k)
                        {
                            for(int p=0; p<K; ++p)
                                cout << img_grad_w_conv[i*C*K*K + j*K*K + k*K + p] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;

               //cin >> k1;


                cout << "Grad Bias : " << i_c << endl;
                for(int i=0; i<this->convs[i_c].get_n_kernels(); ++i)
                {
                    cout << img_grad_b_conv[i] << " ";
                }
                cout << endl;
                

               //cin >> k1;
               */

                
                for(int j=this->n_capas_conv-2; j>=1; j--)
                {
                    // Capa MaxPool 
                    img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[j];
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j];
                    img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[j];
                    this->plms[j].backPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);
                    
                    /*
                    cout << "Output MaxPool " << j << endl;
                    for(int i_=0; i_<this->plms[j].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j].get_W_out(); k_++)
                                cout << img_plms_out[i_*this->plms[j].get_H_out()*this->plms[j].get_W_out() + j_*this->plms[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "PLM_in_copys b: " << j << endl;
                    for(int i_=0; i_<this->plms[j].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j].get_H(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j].get_W(); k_++)
                                cout << img_plms_in_copy[i_*this->plms[j].get_H()*this->plms[j].get_W() + j_*this->plms[j].get_W() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "Conv Out b: " << j << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;
                    */

                    // Capa convolucional 
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[j-1];
                    img_conv_a = conv_a + img*tam_out_convs + i_conv_out[j];
                    img_grad_w_conv = conv_grads_w + i_w[j];
                    img_grad_b_conv = conv_grads_bias + i_b[j];
                    this->convs[j].backPropagationGEMM(img_plms_out, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                    
                    /*
                    cout << "Conv A: " << j << ", " << this->convs[j].get_n_kernels() << "x" << this->convs[j].get_H_out() << "x" << this->convs[j].get_W_out() << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_a[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "Conv Out b: " << j << endl;
                    for(int i_=0; i_<this->convs[j].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[j].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[j].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[j].get_H_out()*this->convs[j].get_W_out() + j_*this->convs[j].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;


                    cout << "Output MaxPool " << j-1 << endl;
                    for(int i_=0; i_<this->plms[j-1].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[j-1].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->plms[j-1].get_W_out(); k_++)
                                cout << img_plms_out[i_*this->plms[j-1].get_H_out()*this->plms[j-1].get_W_out() + j_*this->plms[j-1].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;


                    cout << "Grad Pesos Conv b: " << j << endl;
                    n_kernels = this->convs[j].get_n_kernels(), K = this->convs[j].get_kernel_fils();
                    C = this->convs[j].get_C();
                    for(int i=0; i<n_kernels; ++i)
                    {
                        for(int j=0; j<C; ++j)
                        {
                            for(int k=0; k<K; ++k)
                            {
                                for(int p=0; p<K; ++p)
                                    cout << img_grad_w_conv[i*C*K*K + j*K*K + k*K + p] << " ";
                                cout << endl;
                            }
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "Grad Bias : " << j << endl;
                    for(int i=0; i<this->convs[j].get_n_kernels(); ++i)
                    {
                        cout << img_grad_b_conv[i] << " ";
                    }
                    cout << endl;

                   //cin >> k1;
                   */

                }
                
                
                if(this->n_capas_conv >1)
                {
                    img_conv_out = convs_outs + img*tam_out_convs + i_conv_out[0];
                    img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[0];
                    img_plms_in_copy = plms_in_copys + img*tam_in_pools + i_plm_in[0];
                    this->plms[0].backPropagationGPU(img_conv_out, img_plms_out, img_plms_in_copy);

                    /*
                    cout << "Output MaxPool " << 0 << endl;
                    for(int i_=0; i_<this->plms[0].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[0].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->plms[0].get_W_out(); k_++)
                                cout << img_plms_out[i_*this->plms[0].get_H_out()*this->plms[0].get_W_out() + j_*this->plms[0].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "PLM_in_copys b: " << 0 << endl;
                    for(int i_=0; i_<this->plms[0].get_C(); i_++)
                    {
                        for(int j_=0; j_<this->plms[0].get_H(); j_++)
                        {
                            for(int k_=0; k_<this->plms[0].get_W(); k_++)
                                cout << img_plms_in_copy[i_*this->plms[0].get_H()*this->plms[0].get_W() + j_*this->plms[0].get_W() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "Conv Out b: " << 0 << endl;
                    for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    //cin >> k1;
                    */

                    img_conv_a = conv_a + img*tam_out_convs + i_conv_out[0];
                    img_grad_w_conv = conv_grads_w + i_w[0];
                    img_grad_b_conv = conv_grads_bias + i_b[0];
                    this->convs[0].backPropagationGEMM(img_in, img_conv_out, img_conv_a, img_grad_w_conv, img_grad_b_conv);
                    


                    /*
                    cout << "Conv A: " << 0 << ", " << this->convs[0].get_n_kernels() << "x" << this->convs[0].get_H_out() << "x" << this->convs[0].get_W_out() << endl;
                    for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                                cout << img_conv_a[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "Conv Out b: " << 0 << endl;
                    for(int i_=0; i_<this->convs[0].get_n_kernels(); i_++)
                    {
                        for(int j_=0; j_<this->convs[0].get_H_out(); j_++)
                        {
                            for(int k_=0; k_<this->convs[0].get_W_out(); k_++)
                                cout << img_conv_out[i_*this->convs[0].get_H_out()*this->convs[0].get_W_out() + j_*this->convs[0].get_W_out() + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;


                    cout << "Output MaxPool " << 0 << endl;
                    for(int i_=0; i_<C_ini; i_++)
                    {
                        for(int j_=0; j_<H_ini; j_++)
                        {
                            for(int k_=0; k_<W_ini; k_++)
                                cout << this->img_in[i_*H_ini*W_ini + j_*W_ini + k_] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;
                    */


                    //cout << "Grad Pesos Conv b: " << 0 << endl;
                    n_kernels = this->convs[0].get_n_kernels(), K = this->convs[0].get_kernel_fils();
                    C = this->convs[0].get_C();
                    /*
                    for(int i=0; i<n_kernels; ++i)
                    {
                        for(int j=0; j<C; ++j)
                        {
                            for(int k=0; k<K; ++k)
                            {
                                for(int p=0; p<K; ++p)
                                    cout << img_grad_w_conv[i*C*K*K + j*K*K + k*K + p] << " ";
                                cout << endl;
                            }
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;

                    cout << "Grad Bias : " << 0 << endl;
                    for(int i=0; i<this->convs[0].get_n_kernels(); ++i)
                    {
                        cout << img_grad_b_conv[i] << " ";
                    }
                    cout << endl;

                   //cin >> k1;
                    */
                }
                
            }
            
            
            // ----------------------------------------------
            // Pesos de las capas convolucionales
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media de los gradientes respecto a cada parámetro
            for(int i_=0; i_<tam_kernels_conv; i_++)
                conv_grads_w[i_] /= tam_batches[i];
            
            // ----------------------------------------------
            // Bias o Sesgos de las capas convolucionales
            // ----------------------------------------------
            // ----------------------------------------------
            // Realizar la media
            for(int i_=0; i_<n_bias_conv; i_++)
                conv_grads_bias[i_] /= tam_batches[i];
            
            /*
            cout << "Grad Pesos Conv b: " << 0 << endl;
            for(int r=0; r<n_capas_conv; r++)
            {
                int n_kernels = this->convs[r].get_n_kernels(), K = this->convs[r].get_kernel_fils();
                C = this->convs[r].get_C();
                for(int i=0; i<n_kernels; ++i)
                {
                    for(int j=0; j<C; ++j)
                    {
                        for(int k=0; k<K; ++k)
                        {
                            for(int p=0; p<K; ++p)
                                cout << conv_grads_w[i_w[r] + i*C*K*K + j*K*K + k*K + p] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;
            }
            cout << endl;
            
            /*
            cout << "Grad Bias : " << endl;
            for(int r=0; r<n_capas_conv; r++)
            {
                for(int i=0; i<this->convs[r].get_n_kernels(); ++i)
                {
                    cout << conv_grads_bias[i_b[r] + i] << " ";
                }
                cout << endl;
            }
            cout << endl;
            */

           //cin >> k1;
            


            // Actualizar parámetros --------------------------------------------------------------------
            /*
            cout << "Pesos Conv b: " << endl;
            for(int r=0; r<n_capas_conv; r++)
            {
                float *w_ptr = this->convs[r].get_pesos_ptr();
                int n_kernels = this->convs[r].get_n_kernels(), K = this->convs[r].get_kernel_fils();
                C = this->convs[r].get_C();
                for(int i=0; i<n_kernels; ++i)
                {
                    for(int j=0; j<C; ++j)
                    {
                        for(int k=0; k<K; ++k)
                        {
                            for(int p=0; p<K; ++p)
                                cout << w_ptr[i*C*K*K + j*K*K + k*K + p] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;
            }
            cout << endl;
            */

            
            // Actualizar parámetros de capas convolucionales 
            for(int j=0; j<this->n_capas_conv; ++j)
            {
                img_grad_w_conv = conv_grads_w + i_w[j];
                img_grad_b_conv = conv_grads_bias + i_b[j];
                this->convs[j].actualizar_grads_ptr(img_grad_w_conv, img_grad_b_conv);
            }
            
            // Actualizar parámetros de capas convolucionales 
            for(int j=0; j<this->n_capas_conv; ++j)
                this->convs[j].escalar_pesos_ptr(2);
            
            /*
            cout << "Pesos Conv b: " << endl;
            for(int r=0; r<n_capas_conv; r++)
            {
                float *w_ptr = this->convs[r].get_pesos_ptr();
                int n_kernels = this->convs[r].get_n_kernels(), K = this->convs[r].get_kernel_fils();
                C = this->convs[r].get_C();
                for(int i=0; i<n_kernels; ++i)
                {
                    for(int j=0; j<C; ++j)
                    {
                        for(int k=0; k<K; ++k)
                        {
                            for(int p=0; p<K; ++p)
                                cout << w_ptr[i*C*K*K + j*K*K + k*K + p] << " ";
                            cout << endl;
                        }
                        cout << endl;
                    }
                    cout << endl;
                }
                cout << endl;
            }
            cout << endl;
            
            cin >> k1;
            /*
            cout << "Bias : " << endl;
            for(int r=0; r<n_capas_conv; r++)
            {
                float *bias_ptr = this->convs[r].get_bias_ptr();
                for(int i=0; i<this->convs[r].get_n_kernels(); ++i)
                {
                    cout << bias_ptr[i] << " ";
                }
                cout << endl;
            }
            cout << endl;

           // cin >> k1;
            */
            

        }
        
        fin = high_resolution_clock::now();
        duration = duration_cast<seconds>(fin - ini);
        
        cout << "Época: " << ep << ",                                           " << duration.count() << " (s)" << endl;
        
        checkCudaErrors(hipGetLastError());
        evaluar_modelo();
        
        
    }
    //evaluar_modelo_en_test();
   

    // Liberar memoria
    free(grad_x_fully); free(flat_outs_batch); free(conv_grads_bias); free(grads_bias_fully); free(grads_pesos_fully); free(convs_outs); free(plms_outs); free(conv_grads_w);
    free(plms_in_copys); free(conv_a); free(indices); free(batch); free(tam_batches); free(y_batch);
}

void CNN::mostrar_ptr(float *x, int C, int H, int W)
{
    cout << "\nX\n";
    for(int j=0; j<C; j++)
    {
        for(int k=0; k<H; k++)
        {
            for(int p=0; p<W; p++)
                cout << x[j*H*W + k*W + p] << " ";
            cout << endl;
        }
        cout << endl;
    }
    cout << endl;
}


void CNN::prueba()
{
    int C_in = this->convs[0].get_C(), H_in = this->convs[0].get_H(), W_in = this->convs[0].get_W(),
        C_out = this->convs[0].get_n_kernels(), H_out = this->convs[0].get_H_out(), W_out = this->convs[0].get_W_out();
    float * input = (float *)malloc(C_in*H_in*W_in * sizeof(float)),
          * output = (float *)malloc(C_out*H_out*W_out * sizeof(float)),
          * output_a = (float *)malloc(C_out*H_out*W_out * sizeof(float));
    
    Convolutional conv(this->convs[0].get_n_kernels(), this->convs[0].get_kernel_fils(), this->convs[0].get_kernel_cols(), C_in, H_in, W_in, 0.1);
    
    checkCudaErrors(hipGetLastError());
    cout << "Entro\n";
    this->convs[0].forwardPropagationGEMM(input, output, output_a);
    //conv.forwardPropagationGEMM(input, output, output_a);
    checkCudaErrors(hipGetLastError());
    cout << "Salgo\n";

    free(input); free(output); free(output_a);
}

/*
    @brief  Evalúa el modelo sobre los datos de entrenamiento. Las medidas de evaluación son Accuracy y Entropía Cruzada
*/
void CNN::evaluar_modelo()
{
    float acc=0.0,entr=0.0;
    int C, H, W, C_ini, H_ini, W_ini, H_out, W_out;

    int tam_flat_out = this->plms[this->n_capas_conv-1].get_C() * this->plms[this->n_capas_conv-1].get_H_out() * this->plms[this->n_capas_conv-1].get_W_out();
    float *img_flat_out = nullptr;

    
    // Realizar la propagación hacia delante
    for(int img=0; img<this->n_imagenes; ++img)
    {
        // Copiar imagen de entrenamiento en img_in
        C_ini = this->convs[0].get_C();
        H_ini = this->convs[0].get_H();
        W_ini = this->convs[0].get_W();
        for(int i=0; i<C_ini; i++)
            for(int j=0; j<H_ini; j++)
                for(int k=0; k<W_ini; k++)
                    img_in[i*H_ini*W_ini + j*W_ini + k] = train_imgs[i*H_ini*W_ini + j*W_ini + k + img*C_ini*H_ini*W_ini];
        
        
        // Capas convolucionales y maxpool ----------------------------
        for(int i=0; i<this->n_capas_conv; ++i)
        {
            // Capa convolucional 
            this->convs[i].forwardPropagationGEMM(this->img_in, this->img_out, this->conv_a);
            
            // Copiar img_out en img_in
            C = this->convs[i].get_n_kernels();
            H = this->convs[i].get_H_out();
            W = this->convs[i].get_W_out();

            for(int i=0; i<C; i++)
                for(int j=0; j<H; j++)
                    for(int k=0; k<W; k++)
                        this->img_in[i*H*W + j*W + k] = this->img_out[i*H*W + j*W + k];
            
            // Capa MaxPool 
            this->plms[i].forwardPropagationGPU(this->img_in, this->img_out, this->img_in_copy);
            
            // Copiar img_out en img_in
            H = this->plms[i].get_H_out();
            W = this->plms[i].get_W_out();
            for(int i=0; i<C; i++)
                for(int j=0; j<H; j++)
                    for(int k=0; k<W; k++)
                        this->img_in[i*H*W + j*W + k] = this->img_out[i*H*W + j*W + k];
            
        }

        // Capa flatten
        img_flat_out = flat_outs + img*tam_flat_out; 

        C = this->plms[this->n_capas_conv-1].get_C();
        H_out = this->plms[this->n_capas_conv-1].get_H_out();
        W_out = this->plms[this->n_capas_conv-1].get_W_out();
        //img_plms_out = plms_outs + img*tam_out_pools + i_plm_out[this->n_capas_conv-1];

        for(int i_=0; i_<C; i_++)    
            for(int j_=0; j_<H_out; j_++)    
                for(int k_=0; k_<W_out; k_++)
                    img_flat_out[i_*H_out*W_out + j_*W_out + k_] = img_out[i_*H_out*W_out + j_*W_out + k_];
        
    }
    
    /*
    cout << "FLAT_OUT : " << endl;
    for(int i=0; i<this->n_imagenes; i++)
    {
        for(int j=0; j<tam_flat_out; j++)
            cout << flat_outs[i*tam_flat_out + j] << " ";
        cout << endl;
    }
    cout << endl;
    int k1;
    //cin >> k1;
    */
    

    // Cada hebra obtiene el accuracy y la entropía cruzada sobre una porción de imágenes
    acc = (*this->fully).accuracy_ptr(flat_outs, this->train_labels, n_imagenes, a_ptr, z_ptr);
    entr = (*this->fully).cross_entropy_ptr(flat_outs, this->train_labels, n_imagenes, a_ptr, z_ptr);

    // Realizar media y obtener valores finales
    //acc = acc / n_imagenes * 100;
    //entr = -entr / n_imagenes;

    cout << "Accuracy: " << acc << " %,  ";

    cout << "Entropía cruzada: " << entr << ",         " << endl << endl;    
    checkCudaErrors(hipGetLastError());

}

/*
    @brief  Evalúa el modelo sobre los datos de test. Las medidas de evaluación son Accuracy y Entropía Cruzada
*/
void CNN::evaluar_modelo_en_test()
{
    /*
    int n=this->test_imgs.size();
    double t1, t2;
    vector<vector<vector<float>>> img_in, img_out, img_in_copy, conv_a;
    
    vector<float> flat_out; 
    float acc ,entr;

    vector<vector<float>> flat_outs(n);

    // Inicialización de parámetros
    //t1 = omp_get_wtime();
    acc = 0.0;
    entr = 0.0;


    // Popagación hacia delante
    for(int img=0; img<n; img++)
    {
        img_in = this->test_imgs[img];

        // Capas convolucionales y maxpool ----------------------------
        for(int i=0; i<this->n_capas_conv; ++i)
        {
            // Capa convolucional 
            img_out = this->outputs[i*2];
            conv_a = img_out;
            this->convs[i].forwardPropagation(img_in, img_out, conv_a);
            img_in = img_out;

            // Capa MaxPool 
            img_out = this->outputs[i*2+1];
            img_in_copy = img_in;

            int pad_sig = 0;    // Padding de la siguiente capa convolucional
            if(this->n_capas_conv > i+1)
                pad_sig = this->padding[i+1];

            this->plms[i].forwardPropagation(img_in, img_out, img_in_copy, pad_sig);
            img_in = img_out;
        }
        
        // Capa de aplanado
        (*this->flat).forwardPropagation(img_out, flat_out);

        flat_outs[img] = flat_out;
    }
    
    // Cada hebra obtiene el accuracy y la entropía cruzada sobre una porción de imágenes
    acc = (*this->fully).accuracy(flat_outs,this->test_labels);
    entr = (*this->fully).cross_entropy(flat_outs, this->test_labels);

    // Realizar media y obtener valores finales
    acc = acc / n * 100;
    entr = -entr / n;

    //t2 = omp_get_wtime();

    cout << "\n------------- RESULTADOS EN TEST --------------- " << endl;
    cout << "Accuracy: " << acc << " %,  ";


    cout << "Entropía cruzada: " << entr << ",         " << endl << endl;
    //cout << "Entropía cruzada: " << entr << ",         " << t2 - t1 << " (s) " << endl << endl;
    */
}

/*
int main()
{
    //vector<vector<int>> capas_conv = {{3, 3, 3}, {3, 5, 5}}, tams_pool = {{2, 2}, {2, 2}};
    int C=2, H=10, W=10, n_capas_fully = 2, n_capas_conv = 2, n_imagenes = 5, n_clases = 4;
    int *capas_fully = (int *)malloc(n_capas_fully * sizeof(int)),
        *capas_conv = (int *)malloc(n_capas_conv*3 * sizeof(int)),
        *capas_pool = (int *)malloc(n_capas_conv*2 * sizeof(int)),
        *padding = (int *)malloc(n_capas_conv * sizeof(int));
        
    float *X = (float *)malloc(n_imagenes*C*H*W * sizeof(float)),
        *Y = (float *)malloc(n_imagenes*n_clases * sizeof(float));

    float lr = 0.0001;
    int i=0;
    capas_fully[0] = 2;
    capas_fully[1] = n_clases;

    // Primera capa convolucional
    capas_conv[i*3 +0] = 3;      // 4 kernels
    capas_conv[i*3 +1] = 3;      // kernels de 3 filas
    capas_conv[i*3 +2] = 3;      // kernels de 2 columnas

    i = 1;
    // Segunda capa convolucional
    capas_conv[i*3 +0] = 3;      // 7 kernels
    capas_conv[i*3 +1] = 3;      // kernels de 5 filas
    capas_conv[i*3 +2] = 3;      // kernels de 5 columnas

    i=0;
    // Primera capa MaxPool
    capas_pool[i*2 +0] = 2;      // kernels de 2 filas
    capas_pool[i*2 +1] = 2;      // kernels de 2 columnas

    i = 1;
    // Segunda capa MaxPool
    capas_pool[i*2 +0] = 2;      // kernels de 2 filas
    capas_pool[i*2 +1] = 2;      // kernels de 2 columnas
    
    // Padding
    padding[0] = 0;
    padding[1] = 0;

    // Input
    for(int i=0; i<n_imagenes*C*H*W; i++)
        X[i] = i;

    // Etiquetas
    for(int i=0; i<n_imagenes; i++)
        for(int j=0; j<n_clases; j++)
            Y[i*n_clases + j] = 0.0;

    // Poner que todas las imágenes pertecen a la clase 1, por ejemplo
    for(int i=0; i<n_imagenes; i++)
        Y[i*n_clases + 1] = 1.0;

    CNN cnn(capas_conv, n_capas_conv, capas_pool, padding, capas_fully, n_capas_fully, C, H, W, lr);
    //CNN cnn(capas_conv, n_capas_conv, capas_pool, padding, capas_fully, n_capas_fully, C, H-2*padding[0], W-2*padding[0], lr);
    cnn.mostrar_arquitectura();
    cnn.set_train(X, Y, n_imagenes, n_clases, C, H, W);
    //cnn.evaluar_modelo();
    cnn.train(10, 2);

    free(capas_fully); free(capas_conv); free(capas_pool); free(padding);
    return 0;
}
*/