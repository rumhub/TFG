#include <iostream>
#include <chrono>
#include "random"

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime_api.h"
using namespace std;
using namespace std::chrono;

#define BLOCKSIZE 32

/*
    Para multiplicar A(MxK) x B(KxN) se necesita un bloque de (MxN).
    Cada hebra calcula un elemento de la matriz de salida C, multiplicando una fila de A * una columna de B.
    Usa memoria compartida.
    No usar este método, se queda muy rápido sin memoria por usar bloques tan grandes. No puedes
    multiplicar A(50x50) x B(50x50).
*/
__global__ void multiplicarMatrices2(int M, int N, int K, const float *A, const float *B, float *C)
{
    // Memoria compartida dinámica
	extern __shared__ float sdata[];

    // Convertir de índices de hebra a índices de matriz 
  	int iy = threadIdx.y + blockIdx.y * blockDim.y, ix = threadIdx.x + blockIdx.x * blockDim.x, 
        idA = iy*K + ix, idB = iy*N + ix;
    //int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + (threadIdx.y * blockDim.x + threadIdx.x);
    
    float sum = 0.0f;    

    // Punteros a A y B
    float *sA = sdata,
          *sB = sdata + blockDim.y*K;
    
    // Cada hebra carga en memoria compartida un elemento de A y otro de B
    if(iy < M && ix < K)
        sA[idA] = A[idA];

    if(iy < K && ix < N)
        sB[idB] = B[idB];

    // Sincronizar hebras
    __syncthreads();

    /*
    if(tid == 0)
    {
        
        printf(" --------- A --------- \n");
        for(int i=0; i<M; i++)
        {
            for(int j=0; j<K; j++)
                printf("%f ", A[i*K +j]);
            printf("\n");
        }
        

        printf(" --------- sA --------- \n");
        for(int i=0; i<M; i++)
        {
            for(int j=0; j<K; j++)
                printf("%f ", sA[i*K +j]);
            printf("\n");
        }

        /*
        printf(" --------- B --------- \n");
        for(int i=0; i<K; i++)
        {
            for(int j=0; j<N; j++)
                printf("%f ", B[i*N +j]);
            printf("\n");
        }
        

        printf(" --------- sB --------- \n");
        for(int i=0; i<K; i++)
        {
            for(int j=0; j<N; j++)
                printf("%f ", sB[i*N +j]);
            printf("\n");
        }
        
    }
    */
    
    // Multiplicación matricial
    if(iy < M && ix < N)
    {
        // Cada hebra calcula una posición de C (una fila de A * una columna de B)
        for (int i = 0; i < K; i++) 
            sum += sA[iy*K + i] * sB[ix + i*N];

        C[iy*N + ix] = sum;
    }
    //__syncthreads();
    
    /*
    if(tid == 0)
    {
        printf(" --------- C --------- \n");
        for(int i=0; i<M; i++)
        {
            for(int j=0; j<N; j++)
                printf("%f ", C[i*N +j]);
            printf("\n");
        }
    }
    */
  
}

/*
    Emplea tiles. Un tile por bloque. No usa memoria compartida
*/
__global__ void multiplicarMatrices3(int M, int N, int K, const float *A, const float *B, float *C)
{
    // Memoria compartida dinámica
	extern __shared__ float sdata[];

    // Convertir de índices de hebra a índices de matriz 
  	int iy = threadIdx.y + blockIdx.y * blockDim.y, ix = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0f;    
    
    // Multiplicación matricial
    if(iy < M && ix < N && threadIdx.y < blockDim.y && threadIdx.x < blockDim.x)
    {
        // Cada hebra calcula una posición de C (una fila de A * una columna de B)
        for (int i = 0; i < K; i++) 
            sum += A[iy*K + i] * B[ix + i*N];

        C[iy*N + ix] = sum;
    }
    

    /*
    // Mostrar Tiles
    if(iy < M && ix < N && threadIdx.y < blockDim.y && threadIdx.x < blockDim.x)
    {
        C[iy*N + ix] = blockIdx.x + blockIdx.y;
    }
    */
}

/*
    Emplea tiles. Un tile por bloque. Usa memoria compartida
*/
__global__ void multiplicarMatrices4(int M, int N, int K, const float *A, const float *B, float *C)
{
    // Memoria compartida dinámica
	extern __shared__ float sdata[];

    // Convertir de índices de hebra a índices de matriz 
  	int iy = threadIdx.y + blockIdx.y * blockDim.y, ix = threadIdx.x + blockIdx.x * blockDim.x, 
        idA = iy*K + ix, idB = iy*N + ix, id_tile = threadIdx.y * blockDim.x + threadIdx.x, iy_tile_B = iy, ix_tile_A = ix;;
    int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + (threadIdx.y * blockDim.x + threadIdx.x);
    int n_tiles = (K + blockDim.x - 1) / blockDim.x;

    // Punteros a A y B
    float *sA = sdata,
          *sB = sdata + blockDim.x * blockDim.y;

    float sum = 0.0f;    

    int lim = blockDim.x;

    // Si tam_bloque > tam_A
    if(lim > K)
        lim = K;

    /*
        Para multiplicar A(MxK) x B(KxN) hay que multiplicar una fila de A x una columna de B
        Es decir, multiplicar KxK elementos y sumarlos
        Un tile es más pequeño que K -> Dividir K en tiles e iterar sobre ellos
    */
    for(int tile=0; tile < n_tiles; ++tile)
    {
        idA = iy*K + tile * blockDim.x + threadIdx.x;
        idB = (tile * blockDim.x + threadIdx.y)*N + ix;
       
        // Cargar submatrices de A y B en memoria compartida (tamaño tilex x tiley)
        // Cada hebra carga en memoria compartida un elemento de A y otro de B
        (iy < M && tile * blockDim.x + threadIdx.x < K) ? sA[id_tile] = A[idA] : sA[id_tile] = 0.0;
        (tile * blockDim.x + threadIdx.y < K && ix < N) ? sB[id_tile] = B[idB] : sB[id_tile] = 0.0;

        // Sincronizar hebras
        __syncthreads();

        // Realizar multiplicación matricial
        if(iy < M && ix < N)
        {
            // Si última iteración
            if(tile == n_tiles -1)
                lim = K - tile * blockDim.x;

            // Cada hebra calcula una posición de C (una fila de A * una columna de B)
            for (int i = 0; i < lim; i++) 
                sum += sA[threadIdx.y*blockDim.x + i] * sB[threadIdx.x + i*blockDim.x];
        }

        // Sincronizar hebras
        __syncthreads();
    }

    if(iy < M && ix < N)
        C[iy*N + ix] = sum;
}

__global__ void multiplicarMatrices(int M, int N, int K, const float *A, const float *B, float *C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

   if(row < M && col < N)
   {
        // Una hebra hace una fila de A * una col de B
        for (int i = 0; i < K; i++) {
            sum += A[row*K + i] * B[col + i*N];
        }

        C[row*N + col] = sum;
   }    
}

void multiplyMatrices(float* m1, int rows1, int cols1, float* m2, int cols2, float* result) {
    for (int i = 0; i < rows1; i++) 
        for (int j = 0; j < cols2; j++) {
            result[i * cols2 + j] = 0.0f;

            for (int k = 0; k < cols1; k++) 
                result[i * cols2 + j] += m1[i * cols1 + k] * m2[k * cols2 + j];
            
        }
}


bool comprobarResultados(float *C1, float *C2, int M, int N)
{
    bool correcto = true;
    float epsilon = 0.000000001;
    for(int i=0; i<M; i++)
        for(int j=0; j<N; j++)
            if(abs(C1[i*N +j] - C2[i*N +j]) > epsilon)
            {
                correcto = false;
                //cout << C1[i*N +j] << " vs " << C2[i*N +j] << endl;
            }
    
    return correcto;
}

void printMatrix(float* matrix, int h, int w_) {
    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w_; j++) 
            cout << matrix[i * w_ + j] << " ";
        cout << endl;
    }
}


int main()
{
    // A = MxK, B = KxN, C = MxN
    int M = 2000, K=2000, N=2000,
        bytes_A = M*K * sizeof(float),
        bytes_B = K*N * sizeof(float),
        bytes_C = M*N * sizeof(float);

    dim3 block(N, M);
    dim3 grid(ceil( (float)(N + block.x -1) / block.x), ceil((float)(M + block.y -1) / block.y));

    //cout << "Grid de (" << grid.x << "x" << grid.y << ") " << endl;
    //cout << "Cada bloque es de " << block.x << "x" << block.y << endl;
    // Medidas de tiempo
    auto ini = high_resolution_clock::now();
    auto fin = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(fin - ini);
    
    // Reserva memoria en host
    float *h_A = (float*)malloc(bytes_A),
          *h_B = (float*)malloc(bytes_B),
          *C_cpu = (float*)malloc(bytes_C),
          *h_C_gpu_1 = (float*)malloc(bytes_C),
          *h_C_gpu_2 = (float*)malloc(bytes_C),
          *h_C_gpu_3 = (float*)malloc(bytes_C),
          *h_C_gpu_4 = (float*)malloc(bytes_C);

    // Reserva de memoria en device
    float *d_A, *d_B, *d_C_gpu_1, *d_C_gpu_2, *d_C_gpu_3, *d_C_gpu_4;
    hipMalloc((void **) &d_A, bytes_A);
    hipMalloc((void **) &d_B, bytes_B);
    hipMalloc((void **) &d_C_gpu_1, bytes_C);
    hipMalloc((void **) &d_C_gpu_2, bytes_C);
    hipMalloc((void **) &d_C_gpu_3, bytes_C);
    hipMalloc((void **) &d_C_gpu_4, bytes_C);

    // Inicializar las matrices ----------------
    // Inicializar A
    for(int i=0; i<M; i++)
        for(int j=0; j<K; j++)
            h_A[i*K + j] = i+j;
            //h_A[i*K + j] = i;
            //h_A[i*K + j] = (rand() % 100) + 1;

    // Inicializar B
    for(int i=0; i<K; i++)
        for(int j=0; j<N; j++)
            h_B[i*N + j] = i+j;
            //h_B[i*N + j] = j;
            //h_B[i*N + j] = (rand() % 100) + 1;

    // Copiar matrices A y B de CPU a GPU
    hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes_B, hipMemcpyHostToDevice);
    
    // Multiplicar las matrices en CPU
    ini = high_resolution_clock::now();
    multiplyMatrices(h_A, M, K, h_B, N, C_cpu);
    fin = high_resolution_clock::now();
    duration = duration_cast<microseconds>(fin - ini);
    
    //printMatrix(C_cpu, M, N);

    // Mostrar tiempo
    cout << "Tiempo CPU: " << duration.count() << " (us)" << endl;
    
    // ----------------------------------------- Multiplicar las matrices en GPU -----------------------------------------
    /*
    // Método simple -----------------------------------------
    ini = high_resolution_clock::now();
    multiplicarMatrices<<<grid, block>>>(M, N, K, d_A, d_B, d_C_gpu_1);
    cudaDeviceSynchronize();
    fin = high_resolution_clock::now();
    duration = duration_cast<microseconds>(fin - ini);
    
    // Copiar resultados de GPU a CPU
    cudaMemcpy(h_C_gpu_1, d_C_gpu_1, bytes_C, cudaMemcpyDeviceToHost);

    //printMatrix(h_C_gpu_1, M, N);
    
    // Mostrar resultado
    cout << "Tiempo GPU método simple: " << duration.count() << " (us)" << endl;
    

    // Método con memoria compartida -----------------------------------------
    
    size_t smem = (block.x * K + block.y * K) *sizeof(float);
    ini = high_resolution_clock::now();
    multiplicarMatrices2<<<grid, block, smem>>>(M, N, K, d_A, d_B, d_C_gpu_2);
    cudaDeviceSynchronize();
    fin = high_resolution_clock::now();
    duration = duration_cast<microseconds>(fin - ini);
    
    // Copiar resultados de GPU a CPU
    cudaMemcpy(h_C_gpu_2, d_C_gpu_2, bytes_C, cudaMemcpyDeviceToHost);

    //printMatrix(h_C_gpu_2, M, N);

    
    // Mostrar resultado
    cout << "Tiempo GPU método con memoria compartida: " << duration.count() << " (us)" << endl;
    */

    // Método sin memoria compartida con tiles -----------------------------------------
    dim3 block_tile(BLOCKSIZE, BLOCKSIZE);
    dim3 grid_tile(ceil( (float)(N + block_tile.x -1) / block_tile.x), ceil((float)(M + block_tile.y -1) / block_tile.y));
    
    cout << "Grid de (" << grid_tile.x << "x" << grid_tile.y << ") " << endl;
    cout << "Cada bloque es de " << block_tile.x << "x" << block_tile.y << endl;
    ini = high_resolution_clock::now();
    multiplicarMatrices3<<<grid_tile, block_tile>>>(M, N, K, d_A, d_B, d_C_gpu_3);
    hipDeviceSynchronize();
    fin = high_resolution_clock::now();
    duration = duration_cast<microseconds>(fin - ini);
    
    // Copiar resultados de GPU a CPU
    hipMemcpy(h_C_gpu_3, d_C_gpu_3, bytes_C, hipMemcpyDeviceToHost);

    //printMatrix(h_C_gpu_3, M, N);
    
    // Mostrar resultado
    cout << "Tiempo GPU método sin memoria compartida con tiles: " << duration.count() << " (us)" << endl;


    // Método con memoria compartida con tiles -----------------------------------------
    // Cada tile ocupa (BLOCKSIZExBLOCKSIZE), cada bloque almacena en memoria compartida un tile de A y otro de B
    // Es decir, 2 tiles
    size_t smem_tile = (2*block_tile.x * block_tile.y+1) *sizeof(float);
    ini = high_resolution_clock::now();
    multiplicarMatrices4<<<grid_tile, block_tile, smem_tile>>>(M, N, K, d_A, d_B, d_C_gpu_4);
    hipDeviceSynchronize();
    fin = high_resolution_clock::now();
    duration = duration_cast<microseconds>(fin - ini);
    //cudaError_t err = cudaGetLastError();
    //if (err != cudaSuccess) 
    //    printf("Error: %s\n", cudaGetErrorString(err));

    
    // Copiar resultados de GPU a CPU
    hipMemcpy(h_C_gpu_4, d_C_gpu_4, bytes_C, hipMemcpyDeviceToHost);

    //printMatrix(h_C_gpu_4, M, N);
    
    // Mostrar resultado
    cout << "Tiempo GPU método con memoria compartida con tiles: " << duration.count() << " (us)" << endl;

    // Comprobar resultados
    //if(comprobarResultados(C_cpu, h_C_gpu_1, M, N) && comprobarResultados(C_cpu, h_C_gpu_2, M, N) && comprobarResultados(C_cpu, h_C_gpu_3, M, N) && comprobarResultados(C_cpu, h_C_gpu_4, M, N))
    if(comprobarResultados(h_C_gpu_3, h_C_gpu_4, M, N))
        cout << "Todo correcto!" << endl;
    else
        cout << "Hay errores" << endl;
    
    

    // Liberar memoria
    free(h_A); free(h_B); free(C_cpu);
    hipFree(d_A); hipFree(d_B); hipFree(d_C_gpu_1); hipFree(d_C_gpu_2); hipFree(d_C_gpu_3); hipFree(d_C_gpu_4);


    return 0;
}